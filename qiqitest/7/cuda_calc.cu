#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hipblas.h>
#include "common_data.h"

using namespace std;

int totalMemD = 0;
int permanentMemD = 0;			// 与模型无关的内存使用量（只需要程序退出时释放）
int modelMemD = 0;				// 仅和模型相关的内存使用量（重新载入模型时释放）
int degreeMemD = 0;				// 仅和B样条体次数相关的内存使用量（重新设定B样条体、重新载入模型时释放）
int tessMemD = 0;				// 和细分程度相关的内存使用量（重新设定采样点数、重新设定B样条体、重新载入模型时释放）
int viewMemD = 0;				// 显示函数申请的显存量

//ofstream fout("cuda.txt");

void callCudaThreadSynchronize()
{
	hipDeviceSynchronize();
}

/* B 样条体求值所需的矩阵 */
extern float matrix_b_spline_f[185];
static __device__ float matrix_b_spline_d[185];

/* 根据阶数、控制顶点数、左端节点的编号返回相应的 B 样条矩阵（用于 B 样条体求值） */
template <typename T>
__host__ __device__ T *matrixCase(T *matrix_b_spline, int order, int ctrlPointNum, int leftIdx)
{
	if (order == 1)
		return matrix_b_spline;					// MB1
	else if (order == 2)
		return matrix_b_spline + 1;				// MB2
	else if (order == 3)
	{
		if (ctrlPointNum == 3)
			return matrix_b_spline + 5;			// MB30
		else
		{
			if (leftIdx == 2)
				return matrix_b_spline + 14;	// MB31
			else if (leftIdx == ctrlPointNum - 1)
				return matrix_b_spline + 23;	// MB32
			else
				return matrix_b_spline + 32;	// MB33
		}
	}
	else
	{
		if (ctrlPointNum == 4)
			return matrix_b_spline + 41;		// MB40
		else if (ctrlPointNum == 5)
		{
			if (leftIdx == 3)
				return matrix_b_spline + 57;	// MB41
			else
				return matrix_b_spline + 73;	// MB42
		}
		else if (ctrlPointNum == 6)
		{
			if (leftIdx == 3)
				return matrix_b_spline + 89;	// MB43
			else if (leftIdx == 4)
				return matrix_b_spline + 105;	// MB44
			else
				return matrix_b_spline + 121;	// MB45
		}
		else
		{
			if (leftIdx == 3)
				return matrix_b_spline + 89;	// MB43
			else if (leftIdx == 4)
				return matrix_b_spline + 137;	// MB46
			else if (leftIdx == ctrlPointNum - 2)
				return matrix_b_spline + 153;	// MB47
			else if (leftIdx == ctrlPointNum - 1)
				return matrix_b_spline + 121;	// MB45
			else
				return matrix_b_spline + 169;	// MB48
		}
	}
}

// 便于CPU端调用的一个代理函数
double *matrixCaseHost(double *matrix_b_spline, int order, int ctrlPointNum, int leftIdx)
{
	return matrixCase(matrix_b_spline, order, ctrlPointNum, leftIdx);
}

static __device__ float3 ctrlPointD[15][15][15];	// 原始控制顶点，目前只用于求truth或者FFD结果
static __device__ float knotListD[3 * 20];			// 节点序列

/*
 * 使用矩阵乘法求 B 样条体的值
 * 仅用于 FFD 算法
 */
__device__ float3 BSplineVolumeValueMatrixD(float u, float v, float w,
											int leftUIdx, int leftVIdx, int leftWIdx,
											int orderU, int orderV, int orderW,
											int ctrlPointNumU, int ctrlPointNumV, int ctrlPointNumW)
{
	float3 result;
	float3 tempCtrlPoint1[4];
	float3 tempCtrlPoint2[4][4];

	float *M, temp[4], mul1[4];

	float tempKnot = knotListD[leftUIdx];
	u = (u - tempKnot) / (knotListD[leftUIdx + 1] - tempKnot);
	tempKnot = knotListD[20 + leftVIdx];
	v = (v - tempKnot) / (knotListD[20 + leftVIdx + 1] - tempKnot);
	tempKnot = knotListD[40 + leftWIdx];
	w = (w - tempKnot) / (knotListD[40 + leftWIdx + 1] - tempKnot);

	// 由三维控制顶点算出二维临时控制顶点
	temp[0] = 1.0f;
	temp[1] = w;
	temp[2] = w * w;
	temp[3] = temp[2] * w;

	M = matrixCase(matrix_b_spline_d, orderW, ctrlPointNumW, leftWIdx);

	for (int i = 0; i < orderW; ++i)
	{
		mul1[i] = 0.0f;
		for (int j = 0; j < orderW; ++j)
		{
			mul1[i] += temp[j] * M[j * orderW + i];
		}
	}
	for (int i = 0; i < orderU; ++i)
	{
		for (int j = 0; j < orderV; ++j)
		{
			tempCtrlPoint2[i][j].x = 0.0f;
			tempCtrlPoint2[i][j].y = 0.0f;
			tempCtrlPoint2[i][j].z = 0.0f;
			for (int k = 0; k < orderW; ++k)
			{
				float3 cp = ctrlPointD[leftUIdx - i][leftVIdx - j][leftWIdx - k];
				tempCtrlPoint2[i][j].x += cp.x * mul1[orderW - 1 - k];
				tempCtrlPoint2[i][j].y += cp.y * mul1[orderW - 1 - k];
				tempCtrlPoint2[i][j].z += cp.z * mul1[orderW - 1 - k];
			}
		}
	}

	// 由二维临时控制顶点算出一维临时控制顶点
	temp[1] = v;
	temp[2] = v * v;
	temp[3] = temp[2] * v;

	M = matrixCase(matrix_b_spline_d, orderV, ctrlPointNumV, leftVIdx);

	for (int i = 0; i < orderV; ++i)
	{
		mul1[i] = 0.0;
		for (int j = 0; j < orderV; ++j)
		{
			mul1[i] += temp[j] * M[j * orderV + i];
		}
	}
	for (int i = 0; i < orderU; ++i)
	{
		tempCtrlPoint1[i].x = 0.0f;
		tempCtrlPoint1[i].y = 0.0f;
		tempCtrlPoint1[i].z = 0.0f;
		for (int j = 0; j < orderV; ++j)
		{
			tempCtrlPoint1[i].x += tempCtrlPoint2[i][j].x * mul1[orderV - 1 - j];
			tempCtrlPoint1[i].y += tempCtrlPoint2[i][j].y * mul1[orderV - 1 - j];
			tempCtrlPoint1[i].z += tempCtrlPoint2[i][j].z * mul1[orderV - 1 - j];
		}
	}

	// 由一维临时控制顶点算出结果
	temp[1] = u;
	temp[2] = u * u;
	temp[3] = temp[2] * u;

	M = matrixCase(matrix_b_spline_d, orderU, ctrlPointNumU, leftUIdx);

	for (int i = 0; i < orderU; ++i)
	{
		mul1[i] = 0.0;
		for (int j = 0; j < orderU; ++j)
		{
			mul1[i] += temp[j] * M[j * orderU + i];
		}
	}
	result.x = 0.0f;
	result.y = 0.0f;
	result.z = 0.0f;
	for (int i = 0; i < orderU; ++i)
	{
		result.x += tempCtrlPoint1[i].x * mul1[orderU - 1 - i];
		result.y += tempCtrlPoint1[i].y * mul1[orderU - 1 - i];
		result.z += tempCtrlPoint1[i].z * mul1[orderU - 1 - i];
	}
	return result;
}

/*
 * kernel，计算三个方向参数分别为 u, v, w 的点的 B 样条体值
 * 仅用于 FFD 算法
 */
__global__ void fromParamToCoordOnePoint(float3 *vertexCoordListD, float3 *vertexParamListD,
										 int vertexCount, int orderU, int orderV, int orderW,
										 int ctrlPointNumU, int ctrlPointNumV, int ctrlPointNumW,
										 int knotIntervalCountU, int knotIntervalCountV, int knotIntervalCountW)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= vertexCount)
		return;

	float3 tempVertexParam = vertexParamListD[idx];
	float u = tempVertexParam.x;
	float v = tempVertexParam.y;
	float w = tempVertexParam.z;

	// 预先将其值设为最大，将末端点归入最后一段
	int leftUIdx, leftVIdx, leftWIdx;
	leftUIdx = orderU - 1 + knotIntervalCountU - 1;
	leftVIdx = orderV - 1 + knotIntervalCountV - 1;
	leftWIdx = orderW - 1 + knotIntervalCountW - 1;

	// 沿 U 方向查找当前点所在的节点区间
	for (int i = orderU - 1; i <= orderU - 1 + knotIntervalCountU - 1; ++i)
	{
		if (u >= knotListD[i] && u < knotListD[i + 1])
		{
			leftUIdx = i;
			break;
		}
	}
	// 沿 V 方向查找当前点所在的节点区间
	for (int j = orderV - 1; j <= orderV - 1 + knotIntervalCountV - 1; ++j)
	{
		if (v >= knotListD[20 + j] && v < knotListD[20 + j + 1])
		{
			leftVIdx = j;
			break;
		}
	}
	// 沿 W 方向查找当前点所在的节点区间
	for (int k = orderW - 1; k <= orderW - 1 + knotIntervalCountW - 1; ++k)
	{
		if (w >= knotListD[40 + k] && w < knotListD[40 + k + 1])
		{
			leftWIdx = k;
			break;
		}
	}
	vertexCoordListD[idx] = BSplineVolumeValueMatrixD(u, v, w, leftUIdx, leftVIdx, leftWIdx,
													  orderU, orderV, orderW,
													  ctrlPointNumU, ctrlPointNumV, ctrlPointNumW);
}

float3 *vertexParamListD = 0;					// 模型顶点参数序列
float3 *vertexCoordListD = 0;					// 模型顶点坐标序列

//float3 *vertexParamListD_teapot = 0;					// 模型顶点参数序列
//float3 *normalParamListD_teapot = 0;					// 模型顶点参数序列
//float3 *vertexCoordListD_teapot = 0;					// 模型顶点参数序列
int vertexCount_teapot;

int order[3], ctrlPointNum[3], knotIntervalCount[3], knotCount[3];		// 三个方向的阶数、控制顶点数、节点区间数、节点数
float knotList[3][20];														// 三个方向的节点向量
float3 ctrlPoint[15][15][15];												// B样条体的控制顶点

/*
 * 根据所有顶点的参数，计算出相应的 B 样条体值
 * 仅用于 FFD 算法
 */
void fromParamToCoordD(CommonData *commonData)
{
	int vertexCount = commonData->vertexCount();
	int threadCount = commonData->ffdThreadCount();
	fromParamToCoordOnePoint<<<vertexCount / threadCount + 1, threadCount>>>(
													vertexCoordListD, vertexParamListD,
													vertexCount, order[U], order[V], order[W],
													ctrlPointNum[U], ctrlPointNum[V], ctrlPointNum[W],
													knotIntervalCount[U], knotIntervalCount[V], knotIntervalCount[W]);
	float3 *vertexCoordList = new float3[vertexCount];
	hipMemcpy(vertexCoordList, vertexCoordListD, sizeof(float3) * vertexCount, hipMemcpyDeviceToHost);
	for (int i = 0; i < vertexCount; ++i)
		commonData->setVertexCoord(i, vertexCoordList[i].x, vertexCoordList[i].y, vertexCoordList[i].z);
	delete []vertexCoordList;
}

/*------------------------------------------------------- 上面是FFD算法部分  ---------------------------------------------------------*/
/*------------------------------------------------------- 下面是AFFD算法部分 ---------------------------------------------------------*/

/* 把数字a转换成一个逗号分节的string */
string longNumber(int a)
{
	string result;
	do
	{
		ostringstream oss;

		int remainder = a % 1000;
		if (a >= 1000)
		{
			if (remainder < 10)
				oss << "00" << remainder;
			else if (remainder >= 10 && remainder < 100)
				oss << "0" << remainder;
			else 
				oss << remainder;
		}
		else
			oss << remainder;

		if (result.size() == 0)
			result = oss.str();
		else
			result = oss.str() + "," + result;

		a /= 1000;
	}while(a > 0);

	return result;
}

/* 打印显存使用量 */
void printMemD(const char *file, const char *function, int line, int memSize, string info)
{
	/* 只取文件名部分，路径舍弃 */
	string fileName(file);
	int lastSlashPos = fileName.rfind('/');
	fileName = fileName.substr(lastSlashPos + 1, fileName.size());

/*#define PRINT_MEM*/
#ifdef PRINT_MEM
	/*作废totalMemD += memSize;*/
	cout << info << "\n"
		 << "\t文件" << fileName << "，函数" << function << ", 第" << line << "行，申请显存" << longNumber(memSize) << "字节, "
		 << "目前累计使用显存" << longNumber(permanentMemD + modelMemD + degreeMemD + tessMemD + viewMemD) << "字节\n"
		 << "\t其中permanent = " << longNumber(permanentMemD) << ", model = " << longNumber(modelMemD)
		 << ", degreeMemD = " << longNumber(degreeMemD) << ", tessMemD = " << longNumber(tessMemD)
		 << ", view = " << longNumber(viewMemD) << endl;
#endif
}

void printCudaError(const char *file, const char *function, int line)
{
	/* 只取文件名部分，路径舍弃 */
	string fileName(file);
	int lastSlashPos = fileName.rfind('/');
	fileName = fileName.substr(lastSlashPos + 1, fileName.size());

	hipError_t cymError = hipGetLastError();
	if (cymError)
		cout << fileName << "第" << line << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;
}

__host__ __device__ inline const float3 operator+(const float3 &a, const float3 &b)
{
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__host__ __device__ inline const float3 operator-(const float3 &a, const float3 &b)
{
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__host__ __device__ inline const float3 operator-(const float3 &a)
{
	return make_float3(-a.x, -a.y, -a.z);
}

__host__ __device__ inline const float3 operator*(float a, const float3 &b)
{
	return make_float3(a * b.x, a * b.y, a * b.z);
}

__host__ __device__ inline const float3 operator*(const float3 &a, float b)
{
	return make_float3(a.x * b, a.y * b, a.z * b);
}

__host__ __device__ inline const float3 operator/(const float3 &a, float b)
{
	return make_float3(a.x / b, a.y / b, a.z / b);
}

__host__ __device__ inline float operator*(const float3 &a, const float3 &b)
{
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float3 cross(const float3 &a, const float3 &b)
{
	return make_float3(a.y * b.z - a.z * b.y,
					   a.z * b.x - a.x * b.z,
					   a.x * b.y - a.y * b.x);
}

__host__ __device__ inline void operator*=(float3 &a, float b)
{
	a.x *= b;
	a.y *= b;
	a.z *= b;
}

__host__ __device__ inline void operator/=(float3 &a, float b)
{
	a.x /= b;
	a.y /= b;
	a.z /= b;
}

__host__ __device__ inline void operator+=(float3 &a, const float3 &b)
{
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
}

__host__ __device__ inline void operator-=(float3 &a, const float3 &b)
{
	a.x -= b.x;
	a.y -= b.y;
	a.z -= b.z;
}

__device__ inline float length(const float3 &v)
{
	return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ inline void normalize(float3 &v)
{
	float length_inverse = 1.0 / length(v);
	v *= length_inverse;
}

hipblasHandle_t cublas_handle = 0;

/* 将 B 样条矩阵载入显存 */
void loadMatrixBSplineD()
{
	hipMemcpyToSymbol(HIP_SYMBOL(matrix_b_spline_d), matrix_b_spline_f, sizeof(float) * 185);

	hipblasCreate(&cublas_handle);
}

static __device__ float3 newCtrlPointD[15][15][15][4][4][4];	// 使用皮本上的优化算法之后新生成的控制顶点，每个节点盒都有一个4x4x4的控制顶点

/* 
 * 计算采样点值的优化算法，事先对每个节点盒分别计算B样条体的控制顶点乘以Mu, Mv, Mw的结果并存储
 * 本函数就是进行这个计算，算法具体思路可以看皮本
 */
__global__ void calcNewCtrlPointD(int order_u, int order_v, int order_w,
								  int ctrlPointNum_u, int ctrlPointNum_v, int ctrlPointNum_w)
{
	int ii = blockIdx.x;
	int jj = blockIdx.y;
	int kk = blockIdx.z;

	int leftUIdx = ii + order_u - 1;
	int leftVIdx = jj + order_v - 1;
	int leftWIdx = kk + order_w - 1;
	float *Mu = matrixCase(matrix_b_spline_d, order_u, ctrlPointNum_u, leftUIdx);
	float *Mv = matrixCase(matrix_b_spline_d, order_v, ctrlPointNum_v, leftVIdx);
	float *Mw = matrixCase(matrix_b_spline_d, order_w, ctrlPointNum_w, leftWIdx);

	// 第一个矩阵乘法
	int base_i = leftUIdx - order_u + 1;
	int base_j = leftVIdx - order_v + 1;
	int base_k = leftWIdx - order_w + 1;

	for (int k = 0; k < order_w; ++k)
		for (int i = 0; i < order_u; ++i)
			for (int j = 0; j < order_v; ++j)
			{
				newCtrlPointD[ii][jj][kk][i][j][k] = make_float3(0.0, 0.0, 0.0);
				for (int l = 0; l < order_u; ++l)
				{
					float3 cp = ctrlPointD[base_i + l][base_j + j][base_k + k];
					newCtrlPointD[ii][jj][kk][i][j][k] += Mu[i * order_u + l] * cp;
				}
			}

	// 第二个矩阵乘法
	float3 box[4][4][4];
	for (int i = 0; i < order_u; ++i)
		for (int j = 0; j < order_v; ++j)
			for (int k = 0; k < order_w; ++k)
			{
				box[i][j][k] = make_float3(0.0, 0.0, 0.0);
				for (int l = 0; l < order_v; ++l)
				{
					float3 cp = newCtrlPointD[ii][jj][kk][i][l][k];
					box[i][j][k] += Mv[j * order_v + l] * cp;
				}
			}

	// 第三个矩阵乘法
	for (int j = 0; j < order_v; ++j)
		for (int k = 0; k < order_w; ++k)
			for (int i = 0; i < order_u; ++i)
			{
				newCtrlPointD[ii][jj][kk][i][j][k] = make_float3(0.0, 0.0, 0.0);
				for (int l = 0; l < order_w; ++l)
				{
					float3 cp = box[i][j][l];
							newCtrlPointD[ii][jj][kk][i][j][k] += Mw[k * order_w + l] * cp;
						}
					}
}

/*
 * 将B样条体控制顶点拷贝到显存
 * 另外，将皮本上新算法中的控制顶点拷贝到显存
 */
void copyCtrlPointD(CommonData *commonData)
{
	for (int i = 0; i < ctrlPointNum[U]; ++i)
	{
		for (int j = 0; j < ctrlPointNum[V]; ++j)
		{
			for (int k = 0; k < ctrlPointNum[W]; ++k)
			{
				ctrlPoint[i][j][k].x = (float)commonData->getCtrlPoint(i, j, k).x();
				ctrlPoint[i][j][k].y = (float)commonData->getCtrlPoint(i, j, k).y();
				ctrlPoint[i][j][k].z = (float)commonData->getCtrlPoint(i, j, k).z();
			}
		}
	}
	hipMemcpyToSymbol(HIP_SYMBOL(ctrlPointD), &ctrlPoint[0][0][0], sizeof(float3) * 15 * 15 * 15);
	printCudaError(__FILE__, __FUNCTION__, __LINE__);

	calcNewCtrlPointD<<<dim3(knotIntervalCount[U], knotIntervalCount[V], knotIntervalCount[W]), 1>>>
		(order[U], order[V], order[W], ctrlPointNum[U], ctrlPointNum[V], ctrlPointNum[W]);
	printCudaError(__FILE__, __FUNCTION__, __LINE__);
}

extern float teapot_ctrl_point[3 * 4 * 4 * 36];
static __device__ float3 teapot_ctrl_pointD[4 * 4 * 36];

int u_seg, v_seg;

/* 预计算，将内存中的数据拷贝到相应的显存空间中 */
void preCalcD(CommonData *commonData)
{
	u_seg = commonData->u_seg();
	v_seg = commonData->v_seg();

	float3 temp[16 * 36];
	for (int i = 0; i < 16 * 36; ++i)
	{
		temp[i].x = teapot_ctrl_point[i * 3];
		temp[i].y = teapot_ctrl_point[i * 3 + 1];
		temp[i].z = teapot_ctrl_point[i * 3 + 2];
	}
	hipMemcpyToSymbol(HIP_SYMBOL(teapot_ctrl_pointD), temp, sizeof(float3) * 16 * 36);

	for (int i = 0; i < 3; ++i)
	{
		order[i] = commonData->order(i);
		ctrlPointNum[i] = commonData->ctrlPointCount(i);
		knotIntervalCount[i] = commonData->knotIntervalCount(i);
		knotCount[i] = order[i] + ctrlPointNum[i];
	}
	for (int i = 0; i < 3; ++i)
		for (int j = 0; j < knotCount[i]; ++j)
			knotList[i][j] = (float)commonData->getKnot(i, j);
	hipMemcpyToSymbol(HIP_SYMBOL(knotListD), &knotList[0][0], sizeof(float) * 3 * 20);
	printCudaError(__FILE__, __FUNCTION__, __LINE__);

	int vertexCount = commonData->vertexCount();
	float3 *vertexParamListAlloc = new float3[vertexCount];
	for (int i = 0; i < vertexCount; ++i)
	{
		vertexParamListAlloc[i].x = (float)commonData->vertexParam(i).u();
		vertexParamListAlloc[i].y = (float)commonData->vertexParam(i).v();
		vertexParamListAlloc[i].z = (float)commonData->vertexParam(i).w();
	}
	modelMemD += sizeof(float3) * vertexCount;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float3) * vertexCount, "@原始模型上所有顶点的参数，仅用于FFD");

	hipMalloc((void**)&vertexParamListD, sizeof(float3) * vertexCount);
	printCudaError(__FILE__, __FUNCTION__, __LINE__);

	hipMemcpy(vertexParamListD, vertexParamListAlloc, sizeof(float3) * vertexCount, hipMemcpyHostToDevice);
	printCudaError(__FILE__, __FUNCTION__, __LINE__);

	delete []vertexParamListAlloc;
	vertexParamListAlloc = 0;

	modelMemD += sizeof(float3) * vertexCount;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float3) * vertexCount, "@原始模型上所有顶点的坐标，仅用于FFD");
	hipMalloc((void**)&vertexCoordListD, sizeof(float3) * vertexCount);
	printCudaError(__FILE__, __FUNCTION__, __LINE__);

	copyCtrlPointD(commonData);
	printCudaError(__FILE__, __FUNCTION__, __LINE__);

	// teapot的顶点数据拷贝到显存
	//vertexCount_teapot = commonData->vertexCount_teapot();
	//vertexParamListAlloc = new float3[vertexCount_teapot];
	//for (int i = 0; i < vertexCount_teapot; ++i)
	//{
		//vertexParamListAlloc[i].x = (float)commonData->vertexParam_teapot(i).x();
		//vertexParamListAlloc[i].y = (float)commonData->vertexParam_teapot(i).y();
		//vertexParamListAlloc[i].z = (float)commonData->vertexParam_teapot(i).z();
	//}
	//modelMemD += sizeof(float3) * vertexCount_teapot;
	//hipMalloc((void**)&vertexParamListD_teapot, sizeof(float3) * vertexCount_teapot);
	//printCudaError(__FILE__, __FUNCTION__, __LINE__);
	//hipMemcpy(vertexParamListD_teapot, vertexParamListAlloc, sizeof(float3) * vertexCount_teapot, hipMemcpyHostToDevice);

	//for (int i = 0; i < vertexCount_teapot; ++i)
	//{
		//vertexParamListAlloc[i].x = (float)commonData->normalParam_teapot(i).i();
		//vertexParamListAlloc[i].y = (float)commonData->normalParam_teapot(i).j();
		//vertexParamListAlloc[i].z = (float)commonData->normalParam_teapot(i).k();
	//}
	//hipMalloc((void**)&normalParamListD_teapot, sizeof(float3) * vertexCount_teapot);
	//printCudaError(__FILE__, __FUNCTION__, __LINE__);
	//hipMemcpy(normalParamListD_teapot, vertexParamListAlloc, sizeof(float3) * vertexCount_teapot, hipMemcpyHostToDevice);

	//delete []vertexParamListAlloc;
}

int *matrixFittingIdxD;
float *matrixFittingD;

void loadTriangleMatrixD()
{
	extern int matrixFittingIdx[100];
	hipMalloc((void**)&matrixFittingIdxD, sizeof(int) * 100);
	permanentMemD += sizeof(int) * 100;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(int) * 100, "@拟合矩阵的索引矩阵");
	hipMemcpy(matrixFittingIdxD, matrixFittingIdx, sizeof(int) * 100, hipMemcpyHostToDevice);
	printCudaError(__FILE__, __FUNCTION__, __LINE__);

	extern float matrixFitting[39417];
	hipMalloc((void**)&matrixFittingD, sizeof(float) * 39417);
	permanentMemD += sizeof(float) * 39417;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float) * 39417, "@拟合矩阵");
	hipMemcpy(matrixFittingD, matrixFitting, sizeof(float) * 39417, hipMemcpyHostToDevice);
	printCudaError(__FILE__, __FUNCTION__, __LINE__);
}

struct TriangleD
{
	float3 v[3], n[3], n_adj_origin[3], n_adj[3];
#ifdef LINE
	float3 bary_origin[3];
#endif
	int nc[3];		// nc0, nc1, nc2分别代表v2v0, v0v1, v1v2边的法向数量
	float3 vt[3];
};

TriangleD *triangleListD;
float *sampleValueD, *triangleCtrlPointD;
float3 *sampleValueD_PN;
float *triangleCtrlPointD_PN, *triangleNormalCtrlPointD_PN;
int *triangle_adjacent_tableD;
int degree, degree_lower, triangleCtrlPointNum, triangleCtrlPointNum_lower, triangleNum, constrait_point_num;

int blockSizeStep0 = 128, activeThreadNumStep0, blockNumStep0;
int blockSizeStep1 = 128, activeThreadNumStep1, blockNumStep1;
int blockSizeAdjNormal = 128, activeThreadNumAdjNormal, blockNumAdjNormal;
int blockSizeStep0_PN = 128, blockNumStep0_PN;
#ifdef TRUTH
float *B_1D_truth, *sampleValueD_truth;
int activeThreadNumStep0_truth, blockNumStep0_truth;
#endif

int matrixStartIdxFitting;

__host__ __device__ inline int index2c(int i, int j, int stride)
{
	return j * stride + i;
}

#ifdef DRAW_TRIANGULAR_CTRL_POINTS
float *triangular_ctrl_points;
#endif

void loadTriangleListD(const vector<Triangle> &triangleList, int *triangle_adjacent_table, int deg)
{
	triangleNum = triangleList.size();
	degree = deg;
	/*degree_lower = deg;*/
	degree_lower = 3;
	triangleCtrlPointNum = (degree + 1) * (degree + 2) / 2;
	triangleCtrlPointNum_lower = (degree_lower + 1) * (degree_lower + 2) / 2;
	constrait_point_num = 3 * degree_lower;

	TriangleD *tempTriangleList = new TriangleD[triangleNum];
	for (vector<Triangle>::size_type i = 0; i < triangleNum; ++i)
	{
		for (int j = 0; j < 3; ++j)
		{
			tempTriangleList[i].v[j].x = triangleList[i].v[j].x();
			tempTriangleList[i].v[j].y = triangleList[i].v[j].y();
			tempTriangleList[i].v[j].z = triangleList[i].v[j].z();

			tempTriangleList[i].n[j].x = triangleList[i].n[j].i();
			tempTriangleList[i].n[j].y = triangleList[i].n[j].j();
			tempTriangleList[i].n[j].z = triangleList[i].n[j].k();

			tempTriangleList[i].n_adj_origin[j].x = triangleList[i].n_adj[j].i();
			tempTriangleList[i].n_adj_origin[j].y = triangleList[i].n_adj[j].j();
			tempTriangleList[i].n_adj_origin[j].z = triangleList[i].n_adj[j].k();

			tempTriangleList[i].n_adj[j].x = triangleList[i].n_adj[j].i();
			tempTriangleList[i].n_adj[j].y = triangleList[i].n_adj[j].j();
			tempTriangleList[i].n_adj[j].z = triangleList[i].n_adj[j].k();

#ifdef LINE
			tempTriangleList[i].bary_origin[j].x = triangleList[i].bary_origin[j].x();
			tempTriangleList[i].bary_origin[j].y = triangleList[i].bary_origin[j].y();
			tempTriangleList[i].bary_origin[j].z = triangleList[i].bary_origin[j].z();
#endif

			tempTriangleList[i].nc[j] = triangleList[i].n_count[j];

			tempTriangleList[i].vt[j].x = triangleList[i].vt[j].u();
			tempTriangleList[i].vt[j].y = triangleList[i].vt[j].v();
			tempTriangleList[i].vt[j].z = triangleList[i].vt[j].w();
		}
	}
	hipMalloc((void**)&triangleListD, sizeof(TriangleD) * triangleNum);
	degreeMemD += sizeof(TriangleD) * triangleNum;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(TriangleD) * triangleNum, "@原始模型上所有三角形信息");

	hipMemcpy(triangleListD, tempTriangleList, sizeof(TriangleD) * triangleNum, hipMemcpyHostToDevice);

	delete []tempTriangleList;

	hipMalloc(&sampleValueD, sizeof(float) * (triangleCtrlPointNum + constrait_point_num) * triangleNum * 6);
	degreeMemD += sizeof(float) * (triangleCtrlPointNum + constrait_point_num) * triangleNum * 6;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float) * (triangleCtrlPointNum + constrait_point_num) * triangleNum * 6,
			  "@为了求Bezier曲面片的控制顶点，需要在其上进行采样，结果放在这里。即第二个矩阵乘法用到的矩阵T");

	hipMalloc(&sampleValueD_PN, sizeof(float3) * triangleNum * 3 * 2);
	hipMalloc(&triangleCtrlPointD_PN, sizeof(float) * (1 + 2 + 3 + 4) * triangleNum * 3);
	hipMalloc(&triangleNormalCtrlPointD_PN, sizeof(float) * (1 + 2 + 3) * triangleNum * 3);

	hipMalloc(&triangleCtrlPointD, sizeof(float) * triangleCtrlPointNum_lower * triangleNum * 6);

	hipMalloc(&triangle_adjacent_tableD, sizeof(int) * triangleNum * 3);
	hipMemcpy(triangle_adjacent_tableD, triangle_adjacent_table, sizeof(int) * triangleNum * 3, hipMemcpyHostToDevice);

#ifdef TRUTH
	hipMalloc(&sampleValueD_truth, sizeof(float) * triangleCtrlPointNum * triangleNum * 3);
	degreeMemD += sizeof(float) * triangleCtrlPointNum * triangleNum * 3;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float) * triangleCtrlPointNum * triangleNum * 3,
			  "@为了求精确Bezier曲面片的控制顶点，需要在其上进行采样，结果放在这里。即第二个矩阵乘法用到的矩阵T");

	activeThreadNumStep0_truth = triangleCtrlPointNum * triangleNum;
	blockNumStep0_truth = ceil(static_cast<double>(activeThreadNumStep0_truth) / blockSizeStep0);
#endif

	activeThreadNumStep0 = triangleCtrlPointNum * triangleNum;
	blockNumStep0 = ceil(static_cast<double>(activeThreadNumStep0) / blockSizeStep0);

	activeThreadNumStep1 = constrait_point_num * triangleNum;
	blockNumStep1 = ceil(static_cast<double>(activeThreadNumStep1) / blockSizeStep1);

	activeThreadNumAdjNormal = triangleNum * 3;
	blockNumAdjNormal = ceil(static_cast<double>(activeThreadNumAdjNormal) / blockSizeAdjNormal);

	blockNumStep0_PN = ceil(static_cast<double>(3 * triangleNum) / blockSizeStep0_PN);

#ifdef TRUTH
	extern float matrixTriangle[9][55*55];
	float *temp = new float[triangleCtrlPointNum * triangleCtrlPointNum];
	for (int i = 0; i < triangleCtrlPointNum; ++i)
	{
		for (int j = 0; j < triangleCtrlPointNum; ++j)
		{
			temp[index2c(i, j, triangleCtrlPointNum)] = matrixTriangle[degree - 1][i * triangleCtrlPointNum + j];
		}
	}
	hipMalloc(&B_1D_truth, sizeof(float) * triangleCtrlPointNum * triangleCtrlPointNum);
	degreeMemD += sizeof(float) * triangleCtrlPointNum * triangleCtrlPointNum;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float) * triangleCtrlPointNum * triangleCtrlPointNum, "@第一个矩阵乘法用到的矩阵(B-1)T存放在这里");
	hipMemcpy(B_1D_truth, temp, sizeof(float) * triangleCtrlPointNum * triangleCtrlPointNum, hipMemcpyHostToDevice);
	delete temp;
#endif
	/***************************************************************************/

	extern int matrixFittingIdx[100];
	matrixStartIdxFitting = matrixFittingIdx[degree * 10 + degree_lower];

	cout << "triangleNum = " << triangleNum << endl;
	cout << "degree = " << degree << ", degree_lower = " << degree_lower << ", constrait_point_num = " << constrait_point_num << endl;
	cout << "triangleCtrlPointNum = " << triangleCtrlPointNum << ", triangleCtrlPointNum_lower = " << triangleCtrlPointNum_lower << endl;
	cout << "activeThreadNumStep1 = " << activeThreadNumStep1 << ", blockNumStep1 = " << blockNumStep1 << endl;

#ifdef DRAW_TRIANGULAR_CTRL_POINTS
	triangular_ctrl_points = new float[3 * triangleCtrlPointNum_lower * triangleNum];
#endif
}

double power(double a, int n)
{
	if (n <= 0)
		return 1.0;
	double result = a;
	for (int i = 1; i < n; ++i)
		result *= a;
	return result;
}

int factorial(int n)
{
	int result = 1;
	for (int i = 1; i <= n; ++i)
		result *= i;
	return result;
}

float B(double u, double v, double w, int n, int3 c)
{
	return factorial(n) / factorial(c.x) / factorial(c.y) / factorial(c.z) * power(u, c.x) * power(v, c.y) * power(w, c.z);
}

float *BqD, *BqD_PN, *BBD, *RD;
int *my_to_truth_tableD;
float3 *parameter3D, *parameterND;
#ifdef TRUTH
float *BqD_truth, *BBD_truth, *RD_truth;
#endif
int segmentPerEdge, samplePointPerTriangle;
int blockSizeCopy = 256, activeThreadNumCopy, blockNumCopy;
int *my_to_truth_table;

void generateUVW(int samplePointPerEdge)
{
	segmentPerEdge = samplePointPerEdge - 1;
	samplePointPerTriangle = (samplePointPerEdge + 1) * samplePointPerEdge / 2;

	activeThreadNumCopy = samplePointPerTriangle * triangleNum;
	blockNumCopy = ceil(static_cast<double>(activeThreadNumCopy) / blockSizeCopy);

	double *a = new double[samplePointPerTriangle * 3];
	int idx = 0;
	for (int i = segmentPerEdge; i >= 0; --i)
	{
		for (int j = segmentPerEdge - i; j >= 0; --j)
		{
			int k = segmentPerEdge - i - j;
			a[idx++] = (double)i / segmentPerEdge;
			a[idx++] = (double)j / segmentPerEdge;
			a[idx++] = (double)k / segmentPerEdge;
		}
	}

	float *b = new float[samplePointPerTriangle * triangleCtrlPointNum_lower];
	for (int row = 0; row < samplePointPerTriangle; ++row)
	{
		int idx = 0;
		for (int i = degree_lower; i >= 0; --i)
		{
			for (int j = degree_lower - i; j >= 0; --j)
			{
				int k = degree_lower - i - j;
				double u = a[row * 3 + 0];
				double v = a[row * 3 + 1];
				double w = a[row * 3 + 2];
				b[index2c(row, idx, samplePointPerTriangle)] = B(u, v, w, degree_lower, make_int3(i, j, k));
				++idx;
			}
		}
	}

	float *b_PN = new float[samplePointPerTriangle * 6];
	for (int row = 0; row < samplePointPerTriangle; ++row)
	{
		int idx = 0;
		for (int i = 2; i >= 0; --i)
		{
			for (int j = 2 - i; j >= 0; --j)
			{
				int k = 2 - i - j;
				double u = a[row * 3 + 0];
				double v = a[row * 3 + 1];
				double w = a[row * 3 + 2];
				b_PN[index2c(row, idx, samplePointPerTriangle)] = B(u, v, w, 2, make_int3(i, j, k));
				++idx;
			}
		}
	}

	hipMalloc(&BqD, sizeof(float) * samplePointPerTriangle * triangleCtrlPointNum_lower);
	tessMemD += sizeof(float) * samplePointPerTriangle * triangleCtrlPointNum_lower;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float) * samplePointPerTriangle * triangleCtrlPointNum_lower, "@第一个矩阵乘法用到的矩阵Bq存放在这里");
	hipMemcpy(BqD, b, sizeof(float) * samplePointPerTriangle * triangleCtrlPointNum_lower, hipMemcpyHostToDevice);

	hipMalloc(&BqD_PN, sizeof(float) * samplePointPerTriangle * 6);
	tessMemD += sizeof(float) * samplePointPerTriangle * 6;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float) * samplePointPerTriangle * 6, "@第一个矩阵乘法用到的矩阵Bq存放在这里");
	hipMemcpy(BqD_PN, b_PN, sizeof(float) * samplePointPerTriangle * 6, hipMemcpyHostToDevice);

	/***********************************************************************************************************************************/
	hipMalloc(&BBD, sizeof(float) * samplePointPerTriangle * (triangleCtrlPointNum + constrait_point_num));
	tessMemD += sizeof(float) * samplePointPerTriangle * (triangleCtrlPointNum + constrait_point_num);
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float) * samplePointPerTriangle * (triangleCtrlPointNum + constrait_point_num), "@第二个矩阵乘法用到的矩阵BB存放在这里");

	hipMalloc(&RD, sizeof(float) * samplePointPerTriangle * triangleNum * 6);
	tessMemD += sizeof(float) * samplePointPerTriangle * triangleNum * 6;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float) * samplePointPerTriangle * triangleNum * 6, "@第二个矩阵乘法的结果RD存放在这里");

	hipMalloc(&parameter3D, sizeof(float3) * samplePointPerTriangle * triangleNum);
	hipMalloc(&parameterND, sizeof(float3) * samplePointPerTriangle * triangleNum);

	delete []a;
	delete []b;

	hipMalloc(&my_to_truth_tableD, sizeof(int) * samplePointPerTriangle * triangleNum);
	my_to_truth_table = new int[samplePointPerTriangle * triangleNum];
	fill(my_to_truth_table, my_to_truth_table + samplePointPerTriangle * triangleNum, 0);
}

#ifdef TRUTH
void generateUVW_truth(int samplePointPerEdge)
{
	double *a = new double[samplePointPerTriangle * 3];
	int idx = 0;
	for (int i = segmentPerEdge; i >= 0; --i)
	{
		for (int j = segmentPerEdge - i; j >= 0; --j)
		{
			int k = segmentPerEdge - i - j;
			a[idx++] = (double)i / segmentPerEdge;
			a[idx++] = (double)j / segmentPerEdge;
			a[idx++] = (double)k / segmentPerEdge;
		}
	}

	float *b = new float[samplePointPerTriangle * triangleCtrlPointNum * 3];
	for (int row = 0; row < samplePointPerTriangle; ++row)
	{
		int idx = 0;
		for (int i = degree; i >= 0; --i)
		{
			for (int j = degree - i; j >= 0; --j)
			{
				int k = degree - i - j;
				double u = a[row * 3 + 0];
				double v = a[row * 3 + 1];
				double w = a[row * 3 + 2];
				b[index2c(row, idx, samplePointPerTriangle * 3)] = B(u, v, w, degree, make_int3(i, j, k));
				//b[row * triangleCtrlPointNum + idx] = B(u, v, w, degree, make_int3(i, j, k));
				++idx;
			}
		}
	}

	/***********************************************************************************************************************************/

	for (int row = 0; row < samplePointPerTriangle; ++row)
	{
		int idx = 0;
		for (int i = degree; i >= 0; --i)
		{
			for (int j = degree - i; j >= 0; --j)
			{
				int k = degree - i - j;
				double u = a[row * 3 + 0];
				double v = a[row * 3 + 1];
				double w = a[row * 3 + 2];
				b[index2c(row + samplePointPerTriangle, idx, samplePointPerTriangle * 3)] = factorial(degree) / (factorial(i) * factorial(j) * factorial(k)) *
												 (i * power(u, i - 1) * power(v, j) * power(w, k) - k * power(u, i) * power(v, j) * power(w, k - 1));
				++idx;
			}
		}
	}

	/***********************************************************************************************************************************/

	for (int row = 0; row < samplePointPerTriangle; ++row)
	//for (int row = samplePointPerTriangle * 2; row < samplePointPerTriangle * 3; ++row)
	{
		int idx = 0;
		for (int i = degree; i >= 0; --i)
		{
			for (int j = degree - i; j >= 0; --j)
			{
				int k = degree - i - j;
				double u = a[row * 3 + 0];
				double v = a[row * 3 + 1];
				double w = a[row * 3 + 2];
				b[index2c(row + samplePointPerTriangle * 2, idx, samplePointPerTriangle * 3)] = factorial(degree) / (factorial(i) * factorial(j) * factorial(k)) *
												 (j * power(u, i) * power(v, j - 1) * power(w, k) - k * power(u, i) * power(v, j) * power(w, k - 1));
				++idx;
			}
		}
	}
	hipMalloc(&BqD_truth, sizeof(float) * samplePointPerTriangle * triangleCtrlPointNum * 3);
	tessMemD += sizeof(float) * samplePointPerTriangle * triangleCtrlPointNum * 3;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float) * samplePointPerTriangle * triangleCtrlPointNum * 3, "@第一个矩阵乘法用到的矩阵Bq存放在这里");
	hipMemcpy(BqD_truth, b, sizeof(float) * samplePointPerTriangle * triangleCtrlPointNum * 3, hipMemcpyHostToDevice);

	/***********************************************************************************************************************************/
	hipMalloc(&BBD_truth, sizeof(float) * samplePointPerTriangle * triangleCtrlPointNum * 3);
	tessMemD += sizeof(float) * samplePointPerTriangle * triangleCtrlPointNum * 3;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float) * samplePointPerTriangle * triangleCtrlPointNum * 3, "@第二个矩阵乘法用到的矩阵BB存放在这里");

	hipMalloc(&RD_truth, sizeof(float) * samplePointPerTriangle * 3 * triangleNum * 3);
	tessMemD += sizeof(float) * samplePointPerTriangle * 3 * triangleNum * 3;
	printMemD(__FILE__, __FUNCTION__, __LINE__, sizeof(float) * samplePointPerTriangle * 3 * triangleNum * 3, "@第二个矩阵乘法的结果RD存放在这里");

	delete []a;
	delete []b;
}
#endif

/* 
 * 使用矩阵乘法求 B 样条体的值，和上面一个类似函数的区别在于不负责 u、v、w 重新参数化的工作，
 * 而且也不负责求合适的 B 样条矩阵，这两项工作均需调用函数之前完成，参数列表得到简化
 * 目前仅用于求truth
 */
__device__ float3 BSplineVolumeValueMatrixD2(float *Mu, float *Mv, float *Mw,
											 float u, float v, float w, float *shared_array,
											 int leftUIdx, int leftVIdx, int leftWIdx,
											 int orderU, int orderV, int orderW)
{
#define NB		// NB表示使用比较好的算法，如果不define NB，则使用最原始的算法，逻辑也相对清晰
#ifdef NB
	float *mul1 = (float *)shared_array;
	float *mul2 = (float *)&mul1[blockDim.x * 4];
	float *temp = (float *)&mul2[blockDim.x * 4];

	// 由三维控制顶点算出二维临时控制顶点
	temp[3 * threadIdx.x + 0] = w;
	temp[3 * threadIdx.x + 1] = w * w;
	temp[3 * threadIdx.x + 2] = w * w * w;

	for (int i = 0; i < orderW; ++i)
	{
		mul1[4 * threadIdx.x + i] = Mw[i];
		for (int j = 1; j < orderW; ++j)
			mul1[4 * threadIdx.x + i] += temp[3 * threadIdx.x + j - 1] * Mw[j * orderW + i];
	}

	// 由二维临时控制顶点算出一维临时控制顶点
	temp[3 * threadIdx.x + 0] = v;
	temp[3 * threadIdx.x + 1] = v * v;
	temp[3 * threadIdx.x + 2] = v * v * v;

	for (int i = 0; i < orderV; ++i)
	{
		mul2[4 * threadIdx.x + i] = Mv[i];
		for (int j = 1; j < orderV; ++j)
			mul2[4 * threadIdx.x + i] += temp[3 * threadIdx.x + j - 1] * Mv[j * orderV + i];
	}

	float3 tempCtrlPoint2[4];
	float3 tempCtrlPoint1[4];
	for (int i = 0; i < orderU; ++i)
	{
		for (int j = 0; j < orderV; ++j)
		{
			tempCtrlPoint2[j] = make_float3(0.0f, 0.0f, 0.0f);
			for (int k = 0; k < orderW; ++k)
			{
				float3 cp = ctrlPointD[leftUIdx - i][leftVIdx - j][leftWIdx - k];
				tempCtrlPoint2[j] += cp * mul1[4 * threadIdx.x + orderW - 1 - k];
			}
		}
		tempCtrlPoint1[i] = make_float3(0.0f, 0.0f, 0.0f);
		for (int j = 0; j < orderV; ++j)
			tempCtrlPoint1[i] += tempCtrlPoint2[j] * mul2[4 * threadIdx.x + orderV - 1 - j];
	}

	// 由一维临时控制顶点算出结果
	temp[3 * threadIdx.x + 0] = u;
	temp[3 * threadIdx.x + 1] = u * u;
	temp[3 * threadIdx.x + 2] = u * u * u;

	for (int i = 0; i < orderU; ++i)
	{
		mul1[4 * threadIdx.x + i] = Mu[i];
		for (int j = 1; j < orderU; ++j)
			mul1[4 * threadIdx.x + i] += temp[3 * threadIdx.x + j - 1] * Mu[j * orderU + i];
	}
	float3 result = make_float3(0.0f, 0.0f, 0.0f);
	for (int i = 0; i < orderU; ++i)
		result += tempCtrlPoint1[i] * mul1[4 * threadIdx.x + orderU - 1 - i];

	return result;

	/*-------------------------------------------------------------------------------------------------*/

#else

	// 第一个矩阵乘法
	int base_i = leftUIdx - orderU + 1;
	int base_j = leftVIdx - orderV + 1;
	int base_k = leftWIdx - orderW + 1;
	float3 box[4][4][4], temp;
	for (int k = 0; k < orderW; ++k)
		for (int i = 0; i < orderU; ++i)
			for (int j = 0; j < orderV; ++j)
			{
				temp = make_float3(0.0, 0.0, 0.0);
				for (int l = 0; l < orderU; ++l)
				{
					float3 cp = ctrlPointD[base_i + l][base_j + j][base_k + k];
					temp += Mu[i * orderU + l] * cp;
				}
				box[i][j][k] = temp;
			}

	// 第二个矩阵乘法
	float3 box1[4][4][4];
	for (int i = 0; i < orderU; ++i)
		for (int j = 0; j < orderV; ++j)
			for (int k = 0; k < orderW; ++k)
			{
				temp = make_float3(0.0, 0.0, 0.0);
				for (int l = 0; l < orderV; ++l)
				{
					float3 cp = box[i][l][k];
					temp += Mv[j * orderV + l] * cp;
				}
				box1[i][j][k] = temp;
			}

	// 第三个矩阵乘法
	for (int j = 0; j < orderV; ++j)
		for (int k = 0; k < orderW; ++k)
			for (int i = 0; i < orderU; ++i)
			{
				temp = make_float3(0.0, 0.0, 0.0);
				for (int l = 0; l < orderW; ++l)
				{
					float3 cp = box1[i][j][l];
					temp += Mw[k * orderW + l] * cp;
				}
				box[i][j][k] = temp;
			}

	// 由三维控制顶点算出二维临时控制顶点
	float t[4];
	t[0] = 1.0f;
	t[1] = u;
	t[2] = u * u;
	t[3] = t[2] * u;

	float3 cp2D[4][4];
	for (int j = 0; j < orderV; ++j)
		for (int k = 0; k < orderW; ++k)
		{
			cp2D[j][k] = make_float3(0.0f, 0.0f, 0.0f);
			for (int i = 0; i < orderU; ++i)
			{
				cp2D[j][k] += t[i] * box[i][j][k];
			}
		}

	// 由二维临时控制顶点算出一维临时控制顶点
	t[1] = v;
	t[2] = v * v;
	t[3] = t[2] * v;

	float3 cp1D[4];
	for (int k = 0; k < orderW; ++k)
	{
		cp1D[k] = make_float3(0.0f, 0.0f, 0.0f);
		for (int j = 0; j < orderV; ++j)
			cp1D[k] += t[j] * cp2D[j][k];
	}

	// 由一维临时控制顶点算出结果
	t[1] = w;
	t[2] = w * w;
	t[3] = t[2] * w;

	temp = make_float3(0.0f, 0.0f, 0.0f);
	for (int k = 0; k < orderW; ++k)
		temp += t[k] * cp1D[k];

	return temp;
#endif
}

/* 新的合并算法 */
__device__ void BSplineVolumeValueMatrixD_combine(float u, float v, float w, float *shared_array,
											 int i_idx, int j_idx, int k_idx,
											 int orderU, int orderV, int orderW,
											 float3 &f, float3 &fu, float3 &fv)
{
	int base2 = 2 * threadIdx.x;
	int base3 = 3 * threadIdx.x;
	float *tu = &shared_array[base3];
	float *tu_ = &shared_array[blockDim.x * 3 + base2];
	float *tv = &shared_array[blockDim.x * 5 + base3];
	float *tv_ = &shared_array[blockDim.x * 8 + base2];
	float *tw = &shared_array[blockDim.x * 10 + base3];

	tu[0] = u; tu[1] = u * u, tu[2] = u * tu[1];
	tu_[0] = 2 * u; tu_[1] = 3 * tu[1];

	tv[0] = v; tv[1] = v * v; tv[2] = v * tv[1];
	tv_[0] = 2 * v; tv_[1] = 3 * tv[1];

	tw[0] = w; tw[1] = w * w; tw[2] = w * tw[1];

	/************* 将i = 0 提到前面，减少tu和tu_数组的大小 ****************/
	/******** orderU至少是2，所以这里可以将i = 0的情况提到for之外 *********/
	float3 cp2D[4];
	for (int j = 0; j < orderV; ++j)
	{
		cp2D[j] = newCtrlPointD[i_idx][j_idx][k_idx][0][j][0];
		for (int k = 1; k < orderW; ++k)
			cp2D[j] += tw[k - 1] * newCtrlPointD[i_idx][j_idx][k_idx][0][j][k];
	}
	// orderV至少是2，所以这里可以将tv[0] * cp2D[1]提到for之外
	float3 cp1D = cp2D[0] + tv[0] * cp2D[1], cp1Dv = cp2D[1];
	for (int j = 2; j < orderV; ++j)
	{
		cp1D += tv[j - 1] * cp2D[j];
		cp1Dv += tv_[j - 2] * cp2D[j];
	}
	f = cp1D;
	fv = cp1Dv;

	/*************** 将i = 1 提到前面，减少tu_数组的大小 ******************/
	/******** orderU至少是2，所以这里可以将i = 1的情况提到for之外 *********/
	for (int j = 0; j < orderV; ++j)
	{
		cp2D[j] = newCtrlPointD[i_idx][j_idx][k_idx][1][j][0];
		for (int k = 1; k < orderW; ++k)
			cp2D[j] += tw[k - 1] * newCtrlPointD[i_idx][j_idx][k_idx][1][j][k];
	}
	// orderV至少是2，所以这里可以将tv[0] * cp2D[1]提到for之外
	cp1D = cp2D[0] + tv[0] * cp2D[1];
	cp1Dv = cp2D[1];
	for (int j = 2; j < orderV; ++j)
	{
		cp1D += tv[j - 1] * cp2D[j];
		cp1Dv += tv_[j - 2] * cp2D[j];
	}
	f += tu[0] * cp1D;
	fu = cp1D;
	fv += tu[0] * cp1Dv;

	/*********************************************************************/
	for (int i = 2; i < orderU; ++i)
	{
		for (int j = 0; j < orderV; ++j)
		{
			cp2D[j] = newCtrlPointD[i_idx][j_idx][k_idx][i][j][0];
			for (int k = 1; k < orderW; ++k)
				cp2D[j] += tw[k - 1] * newCtrlPointD[i_idx][j_idx][k_idx][i][j][k];
		}
		// orderV至少是2，所以这里可以将tv[0] * cp2D[1]提到for之外
		cp1D = cp2D[0] + tv[0] * cp2D[1];
		cp1Dv = cp2D[1];
		for (int j = 2; j < orderV; ++j)
		{
			cp1D += tv[j - 1] * cp2D[j];
			cp1Dv += tv_[j - 2] * cp2D[j];
		}
		f += tu[i - 1] * cp1D;
		fu += tu_[i - 2] * cp1D;
		fv += tu[i - 1] * cp1Dv;
	}
}

/* 求采样点在u方向的偏导，由优化之后的采样点求值算法改造而来 */
__device__ float3 BSplineVolumeValueMatrixDu(float u, float v, float w, float *shared_array,
											 int i_idx, int j_idx, int k_idx,
											 int orderU, int orderV, int orderW)
{
	float *tu = (float *)shared_array;
	float *tv = (float *)&tu[blockDim.x * 2];
	float *tw = (float *)&tv[blockDim.x * 3];
	int base2 = 2 * threadIdx.x;
	int base3 = 3 * threadIdx.x;

	tu[base2] = 2 * u; tu[base2 + 1] = 3 * u * u;
	tv[base3] = v; tv[base3 + 1] = v * v; tv[base3 + 2] = v * v * v;
	tw[base3] = w; tw[base3 + 1] = w * w, tw[base3 + 2] = w * w * w;

	// 一步完成三维控制顶点->二维临时控制顶点->一维临时控制顶点->结果
	float3 cp2D[4], cp1D, result;
	for (int j = 0; j < orderV; ++j)
	{
		cp2D[j] = newCtrlPointD[i_idx][j_idx][k_idx][1][j][0];
		for (int k = 1; k < orderU; ++k)
			cp2D[j] += tw[base3 + k - 1] * newCtrlPointD[i_idx][j_idx][k_idx][1][j][k];
	}
	cp1D = cp2D[0];
	for (int j = 1; j < orderV; ++j)
		cp1D += tv[base3 + j - 1] * cp2D[j];
	result = cp1D;

	// 为了把tu从[3]缩成[2]，将i=0的情况提到了前面
	for (int i = 2; i < orderU; ++i)
	{
		for (int j = 0; j < orderV; ++j)
		{
			cp2D[j] = newCtrlPointD[i_idx][j_idx][k_idx][i][j][0];
			for (int k = 1; k < orderW; ++k)
				cp2D[j] += tw[base3 + k - 1] * newCtrlPointD[i_idx][j_idx][k_idx][i][j][k];
		}
		cp1D = cp2D[0];
		for (int j = 1; j < orderV; ++j)
			cp1D += tv[base3 + j - 1] * cp2D[j];

		result += tu[base2 + i - 2] * cp1D;
	}
	return result;

	/*-----------------------------------------------------*/

	/*float tu[4], tv[4], tw[4];*/
	/*tu[0] = 0; tu[1] = 1; tu[2] = 2 * u; tu[3] = 3 * u * u;*/
	/*tv[0] = 1; tv[1] = v; tv[2] = v * v; tv[3] = v * v * v;*/
	/*tw[0] = 1; tw[1] = w; tw[2] = w * w, tw[3] = w * w * w;*/

	/*// 一步完成三维控制顶点->二维临时控制顶点->一维临时控制顶点->结果*/
	/*float3 cp2D[4], cp1D, result = make_float3(0.0f, 0.0f, 0.0f);*/
	/*for (int k = 0; k < orderW; ++k)*/
	/*{*/
		/*for (int j = 0; j < orderV; ++j)*/
		/*{*/
			/*cp2D[j] = make_float3(0.0f, 0.0f, 0.0f);*/
			/*for (int i = 0; i < orderU; ++i)*/
				/*cp2D[j] += tu[i] * newCtrlPointD[i_idx][j_idx][k_idx][i][j][k];*/
		/*}*/
		/*cp1D = make_float3(0.0f, 0.0f, 0.0f);*/
		/*for (int j = 0; j < orderV; ++j)*/
			/*cp1D += tv[j] * cp2D[j];*/

		/*result += tw[k] * cp1D;*/
	/*}*/
	/*return result;*/
}

/* 求采样点在v方向的偏导，由优化之后的采样点求值算法改造而来 */
__device__ float3 BSplineVolumeValueMatrixDv(float u, float v, float w, float *shared_array,
											 int i_idx, int j_idx, int k_idx,
											 int orderU, int orderV, int orderW)
{
	float *tu = (float *)shared_array;
	float *tv = (float *)&tu[blockDim.x * 3];
	float *tw = (float *)&tv[blockDim.x * 2];
	int base2 = 2 * threadIdx.x;
	int base3 = 3 * threadIdx.x;
	tu[base3] = u; tu[base3 + 1] = u * u; tu[base3 + 2] = u * u * u;
	tv[base2] = 2 * v; tv[base2 + 1] = 3 * v * v;
	tw[base3] = w; tw[base3 + 1] = w * w, tw[base3 + 2] = w * w * w;

	// 一步完成三维控制顶点->二维临时控制顶点->一维临时控制顶点->结果
	float3 cp2D[4], cp1D, result;
	for (int k = 0; k < orderW; ++k)
	{
		cp2D[k] = newCtrlPointD[i_idx][j_idx][k_idx][0][1][k];
		for (int i = 1; i < orderU; ++i)
			cp2D[k] += tu[base3 + i - 1] * newCtrlPointD[i_idx][j_idx][k_idx][i][1][k];
	}
	cp1D = cp2D[0];
	for (int k = 1; k < orderW; ++k)
		cp1D += tw[base3 + k - 1] * cp2D[k];
	result = cp1D;

	// 为了把tv从[3]缩成[2]，将j=0的情况提到了前面
	for (int j = 2; j < orderV; ++j)
	{
		for (int k = 0; k < orderW; ++k)
		{
			cp2D[k] = newCtrlPointD[i_idx][j_idx][k_idx][0][j][k];
			for (int i = 1; i < orderU; ++i)
				cp2D[k] += tu[base3 + i - 1] * newCtrlPointD[i_idx][j_idx][k_idx][i][j][k];
		}
		cp1D = cp2D[0];
		for (int k = 1; k < orderW; ++k)
			cp1D += tw[base3 + k - 1] * cp2D[k];

		result += tv[base2 + j - 2] * cp1D;
	}
	return result;

	/*-----------------------------------------------------*/

	/*float tu[4], tv[4], tw[4];*/
	/*tu[0] = 1; tu[1] = u; tu[2] = u * u; tu[3] = u * u * u;*/
	/*tv[0] = 0; tv[1] = 1; tv[2] = 2 * v; tv[3] = 3 * v * v;*/
	/*tw[0] = 1; tw[1] = w; tw[2] = w * w, tw[3] = w * w * w;*/

	/*// 一步完成三维控制顶点->二维临时控制顶点->一维临时控制顶点->结果*/
	/*float3 cp2D[4], cp1D, result = make_float3(0.0f, 0.0f, 0.0f);*/
	/*for (int k = 0; k < orderW; ++k)*/
	/*{*/
		/*for (int j = 0; j < orderV; ++j)*/
		/*{*/
			/*cp2D[j] = make_float3(0.0f, 0.0f, 0.0f);*/
			/*for (int i = 0; i < orderU; ++i)*/
				/*cp2D[j] += tu[i] * newCtrlPointD[i_idx][j_idx][k_idx][i][j][k];*/
		/*}*/
		/*cp1D = make_float3(0.0f, 0.0f, 0.0f);*/
		/*for (int j = 0; j < orderV; ++j)*/
			/*cp1D += tv[j] * cp2D[j];*/

		/*result += tw[k] * cp1D;*/
	/*}*/
	/*return result;*/
}

/* 求采样点在w方向的偏导，由优化之后的采样点求值算法改造而来 */
__device__ float3 BSplineVolumeValueMatrixDw(float u, float v, float w, float *shared_array,
											 int i_idx, int j_idx, int k_idx,
											 int orderU, int orderV, int orderW)
{
	int base2 = 2 * threadIdx.x;
	int base3 = 3 * threadIdx.x;
	float *tu = &shared_array[base3];
	float *tv = &shared_array[blockDim.x * 3 + base3];
	float *tw = &shared_array[blockDim.x * 6 + base2];
	tu[0] = u; tu[1] = u * u; tu[2] = u * tu[1];
	tv[0] = v; tv[1] = v * v; tv[2] = v * tv[1];
	tw[0] = 2 * w; tw[1] = 3 * w * w;

	float3 cp2D[4], cp1D, result;
	for (int i = 0; i < orderU; ++i)
	{
		cp2D[i] = newCtrlPointD[i_idx][j_idx][k_idx][i][0][1];
		for (int j = 1; j < orderV; ++j)
			cp2D[i] += tv[j - 1] * newCtrlPointD[i_idx][j_idx][k_idx][i][j][1];
	}
	cp1D = cp2D[0];
	for (int i = 1; i < orderU; ++i)
		cp1D += tu[i - 1] * cp2D[i];
	result = cp1D;

	// 为了把tw从[3]缩成[2]，将k = 1的情况提到了前面
	for (int k = 2; k < orderW; ++k)
	{
		for (int i = 0; i < orderU; ++i)
		{
			cp2D[i] = newCtrlPointD[i_idx][j_idx][k_idx][i][0][k];
			for (int j = 1; j < orderV; ++j)
				cp2D[i] += tv[j - 1] * newCtrlPointD[i_idx][j_idx][k_idx][i][j][k];
		}
		cp1D = cp2D[0];
		for (int i = 1; i < orderU; ++i)
			cp1D += tu[i - 1] * cp2D[i];

		result += tw[k - 2] * cp1D;
	}
	return result;

	/*-----------------------------------------------------*/

	//float tu[4], tv[4], tw[4];
	//tu[0] = 1; tu[1] = u; tu[2] = u * u; tu[3] = u * u * u;
	//tv[0] = 1; tv[1] = v; tv[2] = v * v, tv[3] = v * v * v;
	//tw[0] = 0; tw[1] = 1; tw[2] = 2 * w; tw[3] = 3 * w * w;

	//float3 cp2D[4], cp1D, result = make_float3(0.0f, 0.0f, 0.0f);
	//for (int i = 0; i < orderU; ++i)
	//{
		//for (int j = 0; j < orderV; ++j)
		//{
			//cp2D[j] = make_float3(0.0f, 0.0f, 0.0f);
			//for (int k = 0; k < orderW; ++k)
				//cp2D[j] += tw[k] * newCtrlPointD[i_idx][j_idx][k_idx][i][j][k];
		//}
		//cp1D = make_float3(0.0f, 0.0f, 0.0f);
		//for (int j = 0; j < orderV; ++j)
			//cp1D += tv[j] * cp2D[j];

		//result += tu[i] * cp1D;
	//}
	//return result;

	/*-----------------------------------------------------*/

	// 最原始的算法
	/*float tu[4], tv[4], tw[4];*/
	/*tu[0] = 1; tu[1] = u; tu[2] = u * u; tu[3] = u * u * u;*/
	/*tv[0] = 1; tv[1] = v; tv[2] = v * v, tv[3] = v * v * v;*/
	/*tw[0] = 0; tw[1] = 1; tw[2] = 2 * w; tw[3] = 3 * w * w;*/

	/*float3 cp2D[4], cp1D, result = make_float3(0.0f, 0.0f, 0.0f);*/
	/*for (int k = 0; k < orderW; ++k)*/
	/*{*/
		/*for (int j = 0; j < orderV; ++j)*/
		/*{*/
			/*cp2D[j] = make_float3(0.0f, 0.0f, 0.0f);*/
			/*for (int i = 0; i < orderU; ++i)*/
				/*cp2D[j] += tu[i] * newCtrlPointD[i_idx][j_idx][k_idx][i][j][k];*/
		/*}*/
		/*cp1D = make_float3(0.0f, 0.0f, 0.0f);*/
		/*for (int j = 0; j < orderV; ++j)*/
			/*cp1D += tv[j] * cp2D[j];*/

		/*result += tw[k] * cp1D;*/
	/*}*/
	/*return result;*/
}

__global__ void calcCtrlPoint_PN(TriangleD *triangleListD, int *triangle_adjacent_tableD, float3 *sampleValueD_PN, float *triangleCtrlPointD_PN, float *triangleNormalCtrlPointD_PN, int f, int m_)
{
	int triangleIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (triangleIdx >= f)
		return;

	int adj_face_idx[3];
	adj_face_idx[0] = triangle_adjacent_tableD[triangleIdx * 3];
	adj_face_idx[1] = triangle_adjacent_tableD[triangleIdx * 3 + 1];
	adj_face_idx[2] = triangle_adjacent_tableD[triangleIdx * 3 + 2];

	//int adj_edge_idx[3] = { -1, -1, -1 };		// 实际上应该使用这一句，以此判断有没有相邻三角形
	int adj_edge_idx[3] = { 0, 0, 0 };			// 但是对于某些模型使用上一句会出现内存越界，所以暂且使用这一句，权宜之计
	//bool handle[3] = { false, false, false };	// 在该边有一个以上法向时，是否处理这条边。当这条边有相邻面片才需要处理
	int adj_corner_ctrlpoint_idx[3][2] = { { 0, 2 }, { 1, 0 }, { 2, 1 } };		// 相邻三角形0, 1, 2号边上的控制顶点编号（仅有角点，没有边点）
	for (int i = 0; i < 3; ++i)
		if (adj_face_idx[i] >= 0)
		{
			adj_edge_idx[i] = adj_face_idx[i] & 0x3;
			adj_face_idx[i] = adj_face_idx[i] >> 2;
			//handle[i] = true;
		}
	//printf("edge_id = (%d, %d, %d), face_id = (%d, %d, %d)\n", adj_edge_idx[0], adj_edge_idx[1], adj_edge_idx[2],
																//adj_face_idx[0], adj_face_idx[1], adj_face_idx[2]);

	int n_count[3];
	n_count[0] = triangleListD[triangleIdx].nc[0];
	n_count[1] = triangleListD[triangleIdx].nc[1];
	n_count[2] = triangleListD[triangleIdx].nc[2];

	float *p_x  = &triangleCtrlPointD_PN[m_ * triangleIdx];
	float *p_y  = &triangleCtrlPointD_PN[m_ * (f + triangleIdx)];
	float *p_z  = &triangleCtrlPointD_PN[m_ * (f * 2 + triangleIdx)];

	float3 v0 = sampleValueD_PN[triangleIdx * 3];
	float3 v1 = sampleValueD_PN[triangleIdx * 3 + 1];
	float3 v2 = sampleValueD_PN[triangleIdx * 3 + 2];
	float3 n0 = sampleValueD_PN[(f + triangleIdx) * 3];
	float3 n1 = sampleValueD_PN[(f + triangleIdx) * 3 + 1];
	float3 n2 = sampleValueD_PN[(f + triangleIdx) * 3 + 2];
	normalize(n0);
	normalize(n1);
	normalize(n2);

	/*********************** 计算几何控制顶点 **********************/
	p_x[0] = v0.x; p_y[0] = v0.y; p_z[0] = v0.z; // 控制顶点0
	p_x[6] = v1.x; p_y[6] = v1.y; p_z[6] = v1.z; // 控制顶点6
	p_x[9] = v2.x; p_y[9] = v2.y; p_z[9] = v2.z; // 控制顶点9

	float3 e = make_float3(0.0f, 0.0f, 0.0f);

	float3 v01 = v1 - v0;
	float3 result;
	if (n_count[1] < 2)		// 该条边只有一个法向
	{
		result = (v0 * 2 + v1 - n0 * (v01 * n0)) / 3;
	}
	else
	{
		float3 n_oppo = triangleListD[adj_face_idx[1]].n_adj[adj_corner_ctrlpoint_idx[adj_edge_idx[1]][0]];
		float3 n_ave = cross(n0, n_oppo);
		normalize(n_ave);
		result = v0 + v01 * n_ave / 3 * n_ave;
	}
	e += result;
	p_x[1] = result.x; p_y[1] = result.y; p_z[1] = result.z; // 控制顶点1

	float3 v02 = v2 - v0;
	if (n_count[0] < 2)		// 该条边只有一个法向
	{
		result = (v0 * 2 + v2 - n0 * (v02 * n0)) / 3;
	}
	else
	{
		float3 n_oppo = triangleListD[adj_face_idx[0]].n_adj[adj_corner_ctrlpoint_idx[adj_edge_idx[0]][1]];
		float3 n_ave = cross(n0, n_oppo);
		normalize(n_ave);
		result = v0 + v02 * n_ave / 3 * n_ave;
	}
	e += result;
	p_x[2] = result.x; p_y[2] = result.y; p_z[2] = result.z; // 控制顶点2

	float3 v10 = v0 - v1;
	if (n_count[1] < 2)		// 该条边只有一个法向
	{
		result = (v1 * 2 + v0 - n1 * (v10 * n1)) / 3;
	}
	else
	{
		float3 n_oppo = triangleListD[adj_face_idx[1]].n_adj[adj_corner_ctrlpoint_idx[adj_edge_idx[1]][1]];
		float3 n_ave = cross(n1, n_oppo);
		normalize(n_ave);
		result = v1 + v10 * n_ave / 3 * n_ave;
	}
	e += result;
	p_x[3] = result.x; p_y[3] = result.y; p_z[3] = result.z; // 控制顶点3

	float3 v12 = v2 - v1;
	if (n_count[2] < 2)		// 该条边只有一个法向
	{
		result = (v1 * 2 + v2 - n1 * (v12 * n1)) / 3;
	}
	else
	{
		float3 n_oppo = triangleListD[adj_face_idx[2]].n_adj[adj_corner_ctrlpoint_idx[adj_edge_idx[2]][0]];
		float3 n_ave = cross(n1, n_oppo);
		normalize(n_ave);
		result = v1 + v12 * n_ave / 3 * n_ave;
	}
	e += result;
	p_x[7] = result.x; p_y[7] = result.y; p_z[7] = result.z; // 控制顶点7

	float3 v20 = v0 - v2;
	if (n_count[0] < 2)		// 该条边只有一个法向
	{
		result = (v2 * 2 + v0 - n2 * (v20 * n2)) / 3;
	}
	else
	{
		float3 n_oppo = triangleListD[adj_face_idx[0]].n_adj[adj_corner_ctrlpoint_idx[adj_edge_idx[0]][0]];
		float3 n_ave = cross(n2, n_oppo);
		normalize(n_ave);
		result = v2 + v20 * n_ave / 3 * n_ave;
	}
	e += result;
	p_x[5] = result.x; p_y[5] = result.y; p_z[5] = result.z; // 控制顶点5

	float3 v21 = v1 - v2;
	if (n_count[2] < 2)		// 该条边只有一个法向
	{
		result = (v2 * 2 + v1 - n2 * (v21 * n2)) / 3;
	}
	else
	{
		float3 n_oppo = triangleListD[adj_face_idx[2]].n_adj[adj_corner_ctrlpoint_idx[adj_edge_idx[2]][1]];
		float3 n_ave = cross(n2, n_oppo);
		normalize(n_ave);
		result = v2 + v21 * n_ave / 3 * n_ave;
	}
	e += result;
	p_x[8] = result.x; p_y[8] = result.y; p_z[8] = result.z; // 控制顶点8

	e /= 6;
	float3 v_total = (v0 + v1 + v2) / 3;
	result = e + (e - v_total) / 2;
	p_x[4] = result.x; p_y[4] = result.y; p_z[4] = result.z; // 控制顶点4


	/*********************** 计算法向控制顶点 **********************/
	p_x  = &triangleNormalCtrlPointD_PN[6 * triangleIdx];
	p_y  = &triangleNormalCtrlPointD_PN[6 * (f + triangleIdx)];
	p_z  = &triangleNormalCtrlPointD_PN[6 * (f * 2 + triangleIdx)];

	p_x[0] = n0.x; p_y[0] = n0.y; p_z[0] = n0.z; // 控制顶点0
	p_x[3] = n1.x; p_y[3] = n1.y; p_z[3] = n1.z; // 控制顶点3
	p_x[5] = n2.x; p_y[5] = n2.y; p_z[5] = n2.z; // 控制顶点5

	float value01 = 2 * v01 * (n0 + n1) / (v01 * v01);
	result = n0 + n1 - value01 * v01;
	normalize(result);
	p_x[1] = result.x; p_y[1] = result.y; p_z[1] = result.z; // 控制顶点1

	float value12 = 2 * v12 * (n1 + n2) / (v12 * v12);
	result = n1 + n2 - value12 * v12;
	normalize(result);
	p_x[4] = result.x; p_y[4] = result.y; p_z[4] = result.z; // 控制顶点4

	float value20 = 2 * v20 * (n2 + n0) / (v20 * v20);
	result = n2 + n0 - value20 * v20;
	normalize(result);
	p_x[2] = result.x; p_y[2] = result.y; p_z[2] = result.z; // 控制顶点2
}

__global__ void calcSampleValueThread_PN(TriangleD *triangleListD, float3 *sampleValueD_PN,
									  int f, int n, int orderU, int orderV, int orderW,
									  int ctrlPointNumU, int ctrlPointNumV, int ctrlPointNumW)
{
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (globalIdx >= 3 * f)
		return;
	int triangleIdx = globalIdx / 3;
	TriangleD &triangle = triangleListD[triangleIdx];

	int localIdx = globalIdx % 3;
	float3 vertex = triangle.v[localIdx];
	float u = vertex.x;
	float v = vertex.y;
	float w = vertex.z;

	//float tempFloorFloat = (sqrtf((float)localIdx * 8 + 9) - 3) * 0.5;
	//int floor = rintf(tempFloorFloat);
	//if ((floor * 2 + 3) * (floor * 2 + 3) != localIdx * 8 + 9)
		//floor = ceilf(tempFloorFloat);
	//int room = localIdx - (floor + 1) * floor * 0.5;
	//float3 barycentric_coord;
	//barycentric_coord.x = (float)(n - floor) / n;
	//barycentric_coord.y = (float)(floor - room) / n;
	//barycentric_coord.z = 1.0f - barycentric_coord.x - barycentric_coord.y;

	//float3 v0 = triangle.v[0];
	//float3 v1 = triangle.v[1];
	//float3 v2 = triangle.v[2];

	//// u, v, w 表示经过重心坐标插值之后的采样点的x, y, z分量
	//float u = v0.x * barycentric_coord.x + v1.x * barycentric_coord.y + v2.x * barycentric_coord.z;
	//float v = v0.y * barycentric_coord.x + v1.y * barycentric_coord.y + v2.y * barycentric_coord.z;
	//float w = v0.z * barycentric_coord.x + v1.z * barycentric_coord.y + v2.z * barycentric_coord.z;

	// u, v, w方向节点区间数量
	int knot_interval_count_u = orderU + ctrlPointNumU - (orderU - 1) * 2 - 1;
	int knot_interval_count_v = orderV + ctrlPointNumV - (orderV - 1) * 2 - 1;
	int knot_interval_count_w = orderW + ctrlPointNumW - (orderW - 1) * 2 - 1;

	// 预先将其值设为最大，将末端点归入最后一段 
	int left_idx_u = orderU - 1 + knot_interval_count_u - 1;
	int left_idx_v = orderV - 1 + knot_interval_count_v - 1;
	int left_idx_w = orderW - 1 + knot_interval_count_w - 1;

	// 沿 U 方向查找当前点所在的节点区间 
	for (int ii = orderU - 1; ii <= orderU - 1 + knot_interval_count_u - 1; ++ii)
	{
		if (u >= knotListD[ii] && u < knotListD[ii + 1])
		{
			left_idx_u = ii;
			break;
		}
	}
	// 沿 V 方向查找当前点所在的节点区间 
	for (int jj = orderV - 1; jj <= orderV - 1 + knot_interval_count_v - 1; ++jj)
	{
		if (v >= knotListD[20 + jj] && v < knotListD[20 + jj + 1])
		{
			left_idx_v = jj;
			break;
		}
	}
	// 沿 W 方向查找当前点所在的节点区间 
	for (int kk = orderW - 1; kk <= orderW - 1 + knot_interval_count_w - 1; ++kk)
	{
		if (w >= knotListD[40 + kk] && w < knotListD[40 + kk + 1])
		{
			left_idx_w = kk;
			break;
		}
	}

	float tmpKnot = knotListD[left_idx_u];
	float tmpKnot1 = knotListD[left_idx_u + 1];
	float x_stride = tmpKnot1 - tmpKnot;
	u = (u - tmpKnot) / x_stride;

	tmpKnot = knotListD[20 + left_idx_v];
	tmpKnot1 = knotListD[20 + left_idx_v + 1];
	float y_stride = tmpKnot1 - tmpKnot;
	v = (v - tmpKnot) / y_stride;

	tmpKnot = knotListD[40 + left_idx_w];
	tmpKnot1 = knotListD[40 + left_idx_w + 1];
	float z_stride = tmpKnot1 - tmpKnot;
	w = (w - tmpKnot) / z_stride;

	extern __shared__ float shared_array[];
	// 算出该线程负责的采样点的 B 样条体值
	// fu 表示J_bar矩阵第一列三个元素：偏F_bar_x偏u、偏F_bar_y偏u、偏F_bar_z偏u
	// fv 表示J_bar矩阵第二列三个元素：偏F_bar_x偏v、偏F_bar_y偏v、偏F_bar_z偏v
	float3 result, fu, fv;
	BSplineVolumeValueMatrixD_combine(u, v, w, shared_array,
	left_idx_u - (orderU - 1), left_idx_v - (orderV - 1), left_idx_w - (orderW - 1),
											   orderU, orderV, orderW,
											   result, fu, fv);
	__syncthreads();

	//sampleValueD[index2c(localIdx, triangleIdx		  , 3)] = result.x;
	//sampleValueD[index2c(localIdx, triangleIdx + f	  , 3)] = result.y;
	//sampleValueD[index2c(localIdx, triangleIdx + f * 2, 3)] = result.z;
	sampleValueD_PN[3 * triangleIdx + localIdx].x = result.x;
	sampleValueD_PN[3 * triangleIdx + localIdx].y = result.y;
	sampleValueD_PN[3 * triangleIdx + localIdx].z = result.z;
	//printf("%d: result = (%f, %f, %f)\n", globalIdx, result.x, result.y, result.z);

	//printf("%d: result = (%f, %f, %f)\n", threadIdx.x, result.x, result.y, result.z);

	///////////////////////////////////////////////////////////////////////////////

	// fw 表示J_bar矩阵第三列三个元素：偏F_bar_x偏w、偏F_bar_y偏w、偏F_bar_z偏w
	float3 fw = BSplineVolumeValueMatrixDw(u, v, w, shared_array,
	left_idx_u - (orderU - 1), left_idx_v - (orderV - 1), left_idx_w - (orderW - 1),
										   orderU, orderV, orderW);
	//__syncthreads();

	//v0 = triangle.n[0];
	//v1 = triangle.n[1];
	//v2 = triangle.n[2];

	//// u, v, w 表示经过重心坐标插值之后的法向的x, y, z分量
	//u = v0.x * barycentric_coord.x + v1.x * barycentric_coord.y + v2.x * barycentric_coord.z;
	//v = v0.y * barycentric_coord.x + v1.y * barycentric_coord.y + v2.y * barycentric_coord.z;
	//w = v0.z * barycentric_coord.x + v1.z * barycentric_coord.y + v2.z * barycentric_coord.z;

	vertex = triangle.n[localIdx];
	u = vertex.x;
	v = vertex.y;
	w = vertex.z;

	float3 *sampleNormalD_PN = sampleValueD_PN + 3 * f;
	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第一行三个元素
	float J_bar_star_T_0 = fv.y * fw.z - fw.y * fv.z;
	float J_bar_star_T_1 = fw.y * fu.z - fu.y * fw.z;
	float J_bar_star_T_2 = fu.y * fv.z - fv.y * fu.z;
	//sampleNormalD[index2c(localIdx, triangleIdx, 3)] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;
	sampleNormalD_PN[3 * triangleIdx + localIdx].x = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;

	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第二行三个元素
	J_bar_star_T_0 = fv.z * fw.x - fw.z * fv.x;
	J_bar_star_T_1 = fw.z * fu.x - fu.z * fw.x;
	J_bar_star_T_2 = fu.z * fv.x - fv.z * fu.x;
	sampleNormalD_PN[3 * triangleIdx + localIdx].y = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;

	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第三行三个元素
	J_bar_star_T_0 = fv.x * fw.y - fw.x * fv.y;
	J_bar_star_T_1 = fw.x * fu.y - fu.x * fw.y;
	J_bar_star_T_2 = fu.x * fv.y - fv.x * fu.y;
	sampleNormalD_PN[3 * triangleIdx + localIdx].z = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;
}

__global__ void calcSampleValueThread(TriangleD *triangleListD, float *sampleValueD,
									  int activeThreadNum, int m, int f, int c, int n,
									  int orderU, int orderV, int orderW,
									  int ctrlPointNumU, int ctrlPointNumV, int ctrlPointNumW)
{
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (globalIdx >= activeThreadNum)
		return;
	int triangleIdx = globalIdx / m;
	TriangleD &triangle = triangleListD[triangleIdx];

	int localIdx = globalIdx % m;

	float tempFloorFloat = (sqrtf((float)localIdx * 8 + 9) - 3) * 0.5;
	int floor = rintf(tempFloorFloat);
	if ((floor * 2 + 3) * (floor * 2 + 3) != localIdx * 8 + 9)
		floor = ceilf(tempFloorFloat);
	int room = localIdx - ((floor + 1) * floor >> 1);
	float3 barycentric_coord;
	barycentric_coord.x = (float)(n - floor) / n;
	barycentric_coord.y = (float)(floor - room) / n;
	barycentric_coord.z = 1.0f - barycentric_coord.x - barycentric_coord.y;

	float3 v0 = triangle.v[0];
	float3 v1 = triangle.v[1];
	float3 v2 = triangle.v[2];

	// u, v, w 表示经过重心坐标插值之后的采样点的x, y, z分量
	float u = v0.x * barycentric_coord.x + v1.x * barycentric_coord.y + v2.x * barycentric_coord.z;
	float v = v0.y * barycentric_coord.x + v1.y * barycentric_coord.y + v2.y * barycentric_coord.z;
	float w = v0.z * barycentric_coord.x + v1.z * barycentric_coord.y + v2.z * barycentric_coord.z;

	// u, v, w方向节点区间数量
	int knot_interval_count_u = orderU + ctrlPointNumU - (orderU - 1) * 2 - 1;
	int knot_interval_count_v = orderV + ctrlPointNumV - (orderV - 1) * 2 - 1;
	int knot_interval_count_w = orderW + ctrlPointNumW - (orderW - 1) * 2 - 1;

	// 预先将其值设为最大，将末端点归入最后一段 
	int left_idx_u = orderU - 1 + knot_interval_count_u - 1;
	int left_idx_v = orderV - 1 + knot_interval_count_v - 1;
	int left_idx_w = orderW - 1 + knot_interval_count_w - 1;

	// 沿 U 方向查找当前点所在的节点区间 
	for (int ii = orderU - 1; ii <= orderU - 1 + knot_interval_count_u - 1; ++ii)
	{
		if (u >= knotListD[ii] && u < knotListD[ii + 1])
		{
			left_idx_u = ii;
			break;
		}
	}
	// 沿 V 方向查找当前点所在的节点区间 
	for (int jj = orderV - 1; jj <= orderV - 1 + knot_interval_count_v - 1; ++jj)
	{
		if (v >= knotListD[20 + jj] && v < knotListD[20 + jj + 1])
		{
			left_idx_v = jj;
			break;
		}
	}
	// 沿 W 方向查找当前点所在的节点区间 
	for (int kk = orderW - 1; kk <= orderW - 1 + knot_interval_count_w - 1; ++kk)
	{
		if (w >= knotListD[40 + kk] && w < knotListD[40 + kk + 1])
		{
			left_idx_w = kk;
			break;
		}
	}

	float tmpKnot = knotListD[left_idx_u];
	float tmpKnot1 = knotListD[left_idx_u + 1];
	float x_stride = tmpKnot1 - tmpKnot;
	u = (u - tmpKnot) / x_stride;

	tmpKnot = knotListD[20 + left_idx_v];
	tmpKnot1 = knotListD[20 + left_idx_v + 1];
	float y_stride = tmpKnot1 - tmpKnot;
	v = (v - tmpKnot) / y_stride;

	tmpKnot = knotListD[40 + left_idx_w];
	tmpKnot1 = knotListD[40 + left_idx_w + 1];
	float z_stride = tmpKnot1 - tmpKnot;
	w = (w - tmpKnot) / z_stride;

	extern __shared__ float shared_array[];
	// 算出该线程负责的采样点的 B 样条体值
	// fu 表示J_bar矩阵第一列三个元素：偏F_bar_x偏u、偏F_bar_y偏u、偏F_bar_z偏u
	// fv 表示J_bar矩阵第二列三个元素：偏F_bar_x偏v、偏F_bar_y偏v、偏F_bar_z偏v
	float3 result, fu, fv;
	BSplineVolumeValueMatrixD_combine(u, v, w, shared_array,
	left_idx_u - (orderU - 1), left_idx_v - (orderV - 1), left_idx_w - (orderW - 1),
											   orderU, orderV, orderW,
											   result, fu, fv);
	__syncthreads();

	sampleValueD[index2c(localIdx, triangleIdx		  , m + c)] = result.x;
	sampleValueD[index2c(localIdx, triangleIdx + f	  , m + c)] = result.y;
	sampleValueD[index2c(localIdx, triangleIdx + f * 2, m + c)] = result.z;

	///////////////////////////////////////////////////////////////////////////////

	// fw 表示J_bar矩阵第三列三个元素：偏F_bar_x偏w、偏F_bar_y偏w、偏F_bar_z偏w
	float3 fw = BSplineVolumeValueMatrixDw(u, v, w, shared_array,
	left_idx_u - (orderU - 1), left_idx_v - (orderV - 1), left_idx_w - (orderW - 1),
										   orderU, orderV, orderW);
	//__syncthreads();

	v0 = triangle.n[0];
	v1 = triangle.n[1];
	v2 = triangle.n[2];

	// u, v, w 表示经过重心坐标插值之后的法向的x, y, z分量
	u = v0.x * barycentric_coord.x + v1.x * barycentric_coord.y + v2.x * barycentric_coord.z;
	v = v0.y * barycentric_coord.x + v1.y * barycentric_coord.y + v2.y * barycentric_coord.z;
	w = v0.z * barycentric_coord.x + v1.z * barycentric_coord.y + v2.z * barycentric_coord.z;

	float *sampleNormalD = sampleValueD + 3 * f * (m + c);
	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第一行三个元素
	float J_bar_star_T_0 = fv.y * fw.z - fw.y * fv.z;
	float J_bar_star_T_1 = fw.y * fu.z - fu.y * fw.z;
	float J_bar_star_T_2 = fu.y * fv.z - fv.y * fu.z;
	sampleNormalD[index2c(localIdx, triangleIdx, m + c)] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;

	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第二行三个元素
	J_bar_star_T_0 = fv.z * fw.x - fw.z * fv.x;
	J_bar_star_T_1 = fw.z * fu.x - fu.z * fw.x;
	J_bar_star_T_2 = fu.z * fv.x - fv.z * fu.x;
	sampleNormalD[index2c(localIdx, triangleIdx + f, m + c)] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;

	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第三行三个元素
	J_bar_star_T_0 = fv.x * fw.y - fw.x * fv.y;
	J_bar_star_T_1 = fw.x * fu.y - fu.x * fw.y;
	J_bar_star_T_2 = fu.x * fv.y - fv.x * fu.y;
	sampleNormalD[index2c(localIdx, triangleIdx + f * 2, m + c)] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;
}

__global__ void calcConstraitSampleValueThread(TriangleD *triangleListD, float *sampleValueD,
											   int activeThreadNum, int m, int f, int c, int n_,
											   int orderU, int orderV, int orderW,
											   int ctrlPointNumU, int ctrlPointNumV, int ctrlPointNumW)
{
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (globalIdx >= activeThreadNum)
		return;
	int triangleIdx = globalIdx / c;
	TriangleD &triangle = triangleListD[triangleIdx];

	int localIdx = globalIdx % c;

	int floor = -1, room = -1;
	if (localIdx < 2 * n_ - 1)
	{
		floor = (localIdx + 1) / 2;
		if (localIdx % 2 == 1)
			room = 0;
		else
			room = floor;
	}
	else
	{
		floor = n_;
		room = localIdx - (2 * n_ - 1);
	}
	float3 barycentric_coord;
	barycentric_coord.x = (float)(n_ - floor) / n_;
	barycentric_coord.y = (float)(floor - room) / n_;
	barycentric_coord.z = 1.0f - barycentric_coord.x - barycentric_coord.y;

	float3 v0 = triangle.v[0];
	float3 v1 = triangle.v[1];
	float3 v2 = triangle.v[2];

	// u, v, w 表示经过重心坐标插值之后的采样点的x, y, z分量
	float u = v0.x * barycentric_coord.x + v1.x * barycentric_coord.y + v2.x * barycentric_coord.z;
	float v = v0.y * barycentric_coord.x + v1.y * barycentric_coord.y + v2.y * barycentric_coord.z;
	float w = v0.z * barycentric_coord.x + v1.z * barycentric_coord.y + v2.z * barycentric_coord.z;

	// u, v, w方向节点区间数量
	int knot_interval_count_u = orderU + ctrlPointNumU - (orderU - 1) * 2 - 1;
	int knot_interval_count_v = orderV + ctrlPointNumV - (orderV - 1) * 2 - 1;
	int knot_interval_count_w = orderW + ctrlPointNumW - (orderW - 1) * 2 - 1;

	// 预先将其值设为最大，将末端点归入最后一段 
	int left_idx_u = orderU - 1 + knot_interval_count_u - 1;
	int left_idx_v = orderV - 1 + knot_interval_count_v - 1;
	int left_idx_w = orderW - 1 + knot_interval_count_w - 1;

	// 沿 U 方向查找当前点所在的节点区间 
	for (int ii = orderU - 1; ii <= orderU - 1 + knot_interval_count_u - 1; ++ii)
	{
		if (u >= knotListD[ii] && u < knotListD[ii + 1])
		{
			left_idx_u = ii;
			break;
		}
	}
	// 沿 V 方向查找当前点所在的节点区间 
	for (int jj = orderV - 1; jj <= orderV - 1 + knot_interval_count_v - 1; ++jj)
	{
		if (v >= knotListD[20 + jj] && v < knotListD[20 + jj + 1])
		{
			left_idx_v = jj;
			break;
		}
	}
	// 沿 W 方向查找当前点所在的节点区间 
	for (int kk = orderW - 1; kk <= orderW - 1 + knot_interval_count_w - 1; ++kk)
	{
		if (w >= knotListD[40 + kk] && w < knotListD[40 + kk + 1])
		{
			left_idx_w = kk;
			break;
		}
	}

	float tmpKnot = knotListD[left_idx_u];
	float tmpKnot1 = knotListD[left_idx_u + 1];
	float x_stride = tmpKnot1 - tmpKnot;
	u = (u - tmpKnot) / x_stride;

	tmpKnot = knotListD[20 + left_idx_v];
	tmpKnot1 = knotListD[20 + left_idx_v + 1];
	float y_stride = tmpKnot1 - tmpKnot;
	v = (v - tmpKnot) / y_stride;

	tmpKnot = knotListD[40 + left_idx_w];
	tmpKnot1 = knotListD[40 + left_idx_w + 1];
	float z_stride = tmpKnot1 - tmpKnot;
	w = (w - tmpKnot) / z_stride;

	extern __shared__ float shared_array[];
	// 算出该线程负责的采样点的 B 样条体值f
	// fu 表示J_bar矩阵第一列三个元素：偏F_bar_x偏u、偏F_bar_y偏u、偏F_bar_z偏u
	// fv 表示J_bar矩阵第二列三个元素：偏F_bar_x偏v、偏F_bar_y偏v、偏F_bar_z偏v
	float3 result, fu, fv;
	BSplineVolumeValueMatrixD_combine(u, v, w, shared_array,
	left_idx_u - (orderU - 1), left_idx_v - (orderV - 1), left_idx_w - (orderW - 1),
											   orderU, orderV, orderW,
											   result, fu, fv);
	__syncthreads();

	sampleValueD[index2c(localIdx + m, triangleIdx		  , m + c)] = result.x;
	sampleValueD[index2c(localIdx + m, triangleIdx + f	  , m + c)] = result.y;
	sampleValueD[index2c(localIdx + m, triangleIdx + f * 2, m + c)] = result.z;

	////////////////////////////////////////////////////////////////////////////

	// fw 表示J_bar矩阵第三列三个元素：偏F_bar_x偏w、偏F_bar_y偏w、偏F_bar_z偏w
	float3 fw = BSplineVolumeValueMatrixDw(u, v, w, shared_array,
	left_idx_u - (orderU - 1), left_idx_v - (orderV - 1), left_idx_w - (orderW - 1),
										   orderU, orderV, orderW);
	//__syncthreads();

	v0 = triangle.n[0];
	v1 = triangle.n[1];
	v2 = triangle.n[2];

	// u, v, w 表示经过重心坐标插值之后的法向的x, y, z分量
	u = v0.x * barycentric_coord.x + v1.x * barycentric_coord.y + v2.x * barycentric_coord.z;
	v = v0.y * barycentric_coord.x + v1.y * barycentric_coord.y + v2.y * barycentric_coord.z;
	w = v0.z * barycentric_coord.x + v1.z * barycentric_coord.y + v2.z * barycentric_coord.z;

	float *sampleNormalD = sampleValueD + 3 * f * (m + c);
	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第一行三个元素
	float J_bar_star_T_0 = fv.y * fw.z - fw.y * fv.z;
	float J_bar_star_T_1 = fw.y * fu.z - fu.y * fw.z;
	float J_bar_star_T_2 = fu.y * fv.z - fv.y * fu.z;
	sampleNormalD[index2c(localIdx + m, triangleIdx, m + c)] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;

	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第二行三个元素
	J_bar_star_T_0 = fv.z * fw.x - fw.z * fv.x;
	J_bar_star_T_1 = fw.z * fu.x - fu.z * fw.x;
	J_bar_star_T_2 = fu.z * fv.x - fv.z * fu.x;
	sampleNormalD[index2c(localIdx + m, triangleIdx + f, m + c)] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;

	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第三行三个元素
	J_bar_star_T_0 = fv.x * fw.y - fw.x * fv.y;
	J_bar_star_T_1 = fw.x * fu.y - fu.x * fw.y;
	J_bar_star_T_2 = fu.x * fv.y - fv.x * fu.y;
	sampleNormalD[index2c(localIdx + m, triangleIdx + f * 2, m + c)] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;
}

__global__ void calcAdjustNormal(TriangleD *triangleListD, int f,
								 int orderU, int orderV, int orderW,
								 int ctrlPointNumU, int ctrlPointNumV, int ctrlPointNumW)
{
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	int triangleIdx = globalIdx / 3;
	if (triangleIdx >= f)
		return;
	int i = globalIdx % 3;

	float3 vertex = triangleListD[triangleIdx].v[i];
	float u = vertex.x;
	float v = vertex.y;
	float w = vertex.z;

	// u, v, w方向节点区间数量
	int knot_interval_count_u = orderU + ctrlPointNumU - (orderU - 1) * 2 - 1;
	int knot_interval_count_v = orderV + ctrlPointNumV - (orderV - 1) * 2 - 1;
	int knot_interval_count_w = orderW + ctrlPointNumW - (orderW - 1) * 2 - 1;

	// 预先将其值设为最大，将末端点归入最后一段 
	int left_idx_u = orderU - 1 + knot_interval_count_u - 1;
	int left_idx_v = orderV - 1 + knot_interval_count_v - 1;
	int left_idx_w = orderW - 1 + knot_interval_count_w - 1;

	// 沿 U 方向查找当前点所在的节点区间 
	for (int ii = orderU - 1; ii <= orderU - 1 + knot_interval_count_u - 1; ++ii)
	{
		if (u >= knotListD[ii] && u < knotListD[ii + 1])
		{
			left_idx_u = ii;
			break;
		}
	}
	// 沿 V 方向查找当前点所在的节点区间 
	for (int jj = orderV - 1; jj <= orderV - 1 + knot_interval_count_v - 1; ++jj)
	{
		if (v >= knotListD[20 + jj] && v < knotListD[20 + jj + 1])
		{
			left_idx_v = jj;
			break;
		}
	}
	// 沿 W 方向查找当前点所在的节点区间 
	for (int kk = orderW - 1; kk <= orderW - 1 + knot_interval_count_w - 1; ++kk)
	{
		if (w >= knotListD[40 + kk] && w < knotListD[40 + kk + 1])
		{
			left_idx_w = kk;
			break;
		}
	}
	float tmpKnot = knotListD[left_idx_u];
	float tmpKnot1 = knotListD[left_idx_u + 1];
	float x_stride = tmpKnot1 - tmpKnot;
	u = (u - tmpKnot) / x_stride;

	tmpKnot = knotListD[20 + left_idx_v];
	tmpKnot1 = knotListD[20 + left_idx_v + 1];
	float y_stride = tmpKnot1 - tmpKnot;
	v = (v - tmpKnot) / y_stride;

	tmpKnot = knotListD[40 + left_idx_w];
	tmpKnot1 = knotListD[40 + left_idx_w + 1];
	float z_stride = tmpKnot1 - tmpKnot;
	w = (w - tmpKnot) / z_stride;

	extern __shared__ float shared_array[];

	// fu 表示J_bar矩阵第一列三个元素：偏F_bar_x偏u、偏F_bar_y偏u、偏F_bar_z偏u
	float3 fu = BSplineVolumeValueMatrixDu(u, v, w, shared_array,
	left_idx_u - (orderU - 1), left_idx_v - (orderV - 1), left_idx_w - (orderW - 1),
										   orderU, orderV, orderW);
	__syncthreads();

	// fv 表示J_bar矩阵第二列三个元素：偏F_bar_x偏v、偏F_bar_y偏v、偏F_bar_z偏v
	float3 fv = BSplineVolumeValueMatrixDv(u, v, w, shared_array,
	left_idx_u - (orderU - 1), left_idx_v - (orderV - 1), left_idx_w - (orderW - 1),
										   orderU, orderV, orderW);
	__syncthreads();

	// fw 表示J_bar矩阵第三列三个元素：偏F_bar_x偏w、偏F_bar_y偏w、偏F_bar_z偏w
	float3 fw = BSplineVolumeValueMatrixDw(u, v, w, shared_array,
	left_idx_u - (orderU - 1), left_idx_v - (orderV - 1), left_idx_w - (orderW - 1),
										   orderU, orderV, orderW);
	__syncthreads();

	vertex = triangleListD[triangleIdx].n_adj_origin[i];
	u = vertex.x;
	v = vertex.y;
	w = vertex.z;

	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第一行三个元素
	float J_bar_star_T_0 = fv.y * fw.z - fw.y * fv.z;
	float J_bar_star_T_1 = fw.y * fu.z - fu.y * fw.z;
	float J_bar_star_T_2 = fu.y * fv.z - fv.y * fu.z;
	triangleListD[triangleIdx].n_adj[i].x = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;

	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第二行三个元素
	J_bar_star_T_0 = fv.z * fw.x - fw.z * fv.x;
	J_bar_star_T_1 = fw.z * fu.x - fu.z * fw.x;
	J_bar_star_T_2 = fu.z * fv.x - fv.z * fu.x;
	triangleListD[triangleIdx].n_adj[i].y = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;

	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第三行三个元素
	J_bar_star_T_0 = fv.x * fw.y - fw.x * fv.y;
	J_bar_star_T_1 = fw.x * fu.y - fu.x * fw.y;
	J_bar_star_T_2 = fu.x * fv.y - fv.x * fu.y;
	triangleListD[triangleIdx].n_adj[i].z = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;
}

void calcSampleValue(AlgorithmType algo_type)
{
	if (algo_type == CYM)
	{
		// 计算采样点的值和法向
		//calcSampleValueThread<<<blockNumStep0, blockSizeStep0, sizeof(float) * blockSizeStep0 * 9>>>
		calcSampleValueThread<<<blockNumStep0, blockSizeStep0, sizeof(float) * blockSizeStep0 * 13>>>
											(triangleListD, sampleValueD,
											 activeThreadNumStep0, triangleCtrlPointNum, triangleNum, constrait_point_num,
											 degree, order[U], order[V], order[W],
											 ctrlPointNum[U], ctrlPointNum[V], ctrlPointNum[W]);

		// 计算约束点的值和法向
		//calcConstraitSampleValueThread<<<blockNumStep1, blockSizeStep1, sizeof(float) * blockSizeStep1 * 9>>>
		calcConstraitSampleValueThread<<<blockNumStep1, blockSizeStep1, sizeof(float) * blockSizeStep1 * 13>>>
											(triangleListD, sampleValueD,
											 activeThreadNumStep1, triangleCtrlPointNum, triangleNum, constrait_point_num,
											 degree_lower, order[U], order[V], order[W],
											 ctrlPointNum[U], ctrlPointNum[V], ctrlPointNum[W]);
	}
	else
	{
		calcSampleValueThread_PN<<<blockNumStep0_PN, blockSizeStep0_PN, sizeof(float) * blockSizeStep1 * 13>>>
										(triangleListD, sampleValueD_PN,
										  triangleNum, degree, order[U], order[V], order[W],
										  ctrlPointNum[U], ctrlPointNum[V], ctrlPointNum[W]);
	}
	//hipError_t error = hipGetLastError();
	//if (error != 0)
	//{
		//cout << "第0步出错\t";
		//printf("CUDA error: %s\n", hipGetErrorString(error));
	//}

	//error = hipGetLastError();
	//if (error != 0)
	//{
		//cout << "第一步出错\t";
		//printf("CUDA error: %s\n", hipGetErrorString(error));
	//}
	//float3 *test = new float3[triangleNum * 3];
	//for (int i = 0; i < triangleNum * 3; ++i)
		//test[i] = make_float3(1.0f, 2.0f, 3.0f);
	//hipMemcpy(test, sampleValueD_PN, sizeof(float3) * triangleNum * 3, hipMemcpyDeviceToHost);
	//for (int i = 0; i < triangleNum; ++i)
	//{
		//cout << test[i * 3].x << ", " << test[i * 3].y << ", " << test[i * 3].z << endl;
		//cout << test[i * 3 + 1].x << ", " << test[i * 3 + 1].y << ", " << test[i * 3 + 1].z << endl;
		//cout << test[i * 3 + 2].x << ", " << test[i * 3 + 2].y << ", " << test[i * 3 + 2].z << endl;
		//cout << "==============" << endl;
	//}
	//delete []test;

	//float *test = new float[(triangleCtrlPointNum + constrait_point_num) * triangleNum * 6];
	//hipMemcpy(test, sampleValueD, sizeof(float) * (triangleCtrlPointNum + constrait_point_num) * triangleNum * 6, hipMemcpyDeviceToHost);
	//float *n = test + (triangleCtrlPointNum + constrait_point_num) * triangleNum * 3;
	//for (int i = 0; i < triangleNum; ++i)
	//{
		//cout << "i = " << i << endl;
		//for (int j = 0; j < constrait_point_num; ++j)
		//{
			//cout << "\t" << j << " " << test[i * (triangleCtrlPointNum + constrait_point_num) + triangleCtrlPointNum + j] << " "
				 //<< test[(i + triangleNum) * (triangleCtrlPointNum + constrait_point_num) + triangleCtrlPointNum + j] << " "
				 //<< test[(i + triangleNum * 2) * (triangleCtrlPointNum + constrait_point_num) + triangleCtrlPointNum + j] << "\t";
			//cout << n[i * (triangleCtrlPointNum + constrait_point_num) + triangleCtrlPointNum + j] << " "
				 //<< n[(i + triangleNum) * (triangleCtrlPointNum + constrait_point_num) + triangleCtrlPointNum + j] << " "
				 //<< n[(i + triangleNum * 2) * (triangleCtrlPointNum + constrait_point_num) + triangleCtrlPointNum + j] << endl;
		//}
	//}
	//delete []test;
}

#ifdef TRUTH
__global__ void calcSampleValueThread_truth(TriangleD *triangleListD, float *sampleValueD_truth,
									  int activeThreadNum, int m, int f, int n,
									  int orderU, int orderV, int orderW,
									  int ctrlPointNumU, int ctrlPointNumV, int ctrlPointNumW)
{
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (globalIdx >= activeThreadNum)
		return;

	int triangleIdx = globalIdx / m;

	int localIdx = globalIdx % m;

	float tempFloorFloat = (sqrtf((float)localIdx * 8 + 9) - 3) / 2;
	int floor = rintf(tempFloorFloat);
	if ((floor * 2 + 3) * (floor * 2 + 3) != localIdx * 8 + 9)
		floor = ceilf(tempFloorFloat);
	int room = localIdx - (floor + 1) * floor / 2;
	float3 barycentric_coord;
	barycentric_coord.x = (float)(n - floor) / n;
	barycentric_coord.y = (float)(floor - room) / n;
	barycentric_coord.z = 1.0f - barycentric_coord.x - barycentric_coord.y;

	TriangleD &triangle = triangleListD[triangleIdx];
	float3 v0 = triangle.v[0];
	float3 v1 = triangle.v[1];
	float3 v2 = triangle.v[2];

	float u = v0.x * barycentric_coord.x + v1.x * barycentric_coord.y + v2.x * barycentric_coord.z;
	float v = v0.y * barycentric_coord.x + v1.y * barycentric_coord.y + v2.y * barycentric_coord.z;
	float w = v0.z * barycentric_coord.x + v1.z * barycentric_coord.y + v2.z * barycentric_coord.z;

	int i = (u - knotListD[0]) / (knotListD[orderU] - knotListD[0]);
	int j = (v - knotListD[20 + 0]) / (knotListD[20 + orderV] - knotListD[20 + 0]);
	int k = (w - knotListD[40 + 0]) / (knotListD[40 + orderW] - knotListD[40 + 0]);
	if (i >= ctrlPointNumU + orderU - 2 * (orderU - 1) - 1)
		--i;
	if (j >= ctrlPointNumV + orderV - 2 * (orderV - 1) - 1)
		--j;
	if (k >= ctrlPointNumW + orderW - 2 * (orderW - 1) - 1)
		--k;

	/* 确定此 block 需要的 u、v、w 三个方向的 B 样条矩阵 */
	float *Mu = matrixCase(matrix_b_spline_d, orderU, ctrlPointNumU, i + orderU - 1);
	float *Mv = matrixCase(matrix_b_spline_d, orderV, ctrlPointNumV, j + orderV - 1);
	float *Mw = matrixCase(matrix_b_spline_d, orderW, ctrlPointNumW, k + orderW - 1);

	float tmpKnot = knotListD[i + orderU - 1];
	float tmpKnot1 = knotListD[i + orderU];
	u = (u - tmpKnot) / (tmpKnot1 - tmpKnot);

	tmpKnot = knotListD[20 + j + orderV - 1];
	tmpKnot1 = knotListD[20 + j + orderV];
	v = (v - tmpKnot) / (tmpKnot1 - tmpKnot);

	tmpKnot = knotListD[40 + k + orderW - 1];
	tmpKnot1 = knotListD[40 + k + orderW];
	w = (w - tmpKnot) / (tmpKnot1 - tmpKnot);

	extern __shared__ float shared_array[];
	/* 算出该线程负责的采样点的 B 样条体值 */
	float3 result = BSplineVolumeValueMatrixD2(Mu, Mv, Mw,
											   u, v, w, shared_array,
											   i + orderU - 1, j + orderV - 1, k + orderW - 1,
											   orderU, orderV, orderW);

	sampleValueD_truth[index2c(localIdx, triangleIdx, m)] = result.x;
	sampleValueD_truth[index2c(localIdx, triangleIdx + f, m)] = result.y;
	sampleValueD_truth[index2c(localIdx, triangleIdx + f * 2, m)] = result.z;
}

void calcSampleValue_truth()
{
	calcSampleValueThread_truth<<<blockNumStep0_truth, blockSizeStep0, sizeof(float) * blockSizeStep0 * 11>>>
										(triangleListD, sampleValueD_truth,
										 activeThreadNumStep0_truth, triangleCtrlPointNum, triangleNum,
										 degree, order[U], order[V], order[W],
										 ctrlPointNum[U], ctrlPointNum[V], ctrlPointNum[W]);
}
#endif

/************************************************************************************************************/
#define NEW_MOVE			// 定义这个表示使用最终的move函数，否则使用最原始的PN算法
#ifdef NEW_MOVE
// 调整拟合出来的控制顶点
__global__ void move(TriangleD *triangleListD, float *triangleCtrlPointD, int *triangle_adjacent_tableD,
					 int m_, int f, float center_factor, bool use_pn)
{
	int triangleIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if (triangleIdx >= f)
		return;

	int adj_face_idx[3];
	adj_face_idx[0] = triangle_adjacent_tableD[triangleIdx * 3];
	adj_face_idx[1] = triangle_adjacent_tableD[triangleIdx * 3 + 1];
	adj_face_idx[2] = triangle_adjacent_tableD[triangleIdx * 3 + 2];

	//int adj_edge_idx[3] = { -1, -1, -1 };		// 实际上应该使用这一句，以此判断有没有相邻三角形
	int adj_edge_idx[3] = { 0, 0, 0 };			// 但是对于某些模型使用上一句会出现内存越界，所以暂且使用这一句，权宜之计
	//bool handle[3] = { false, false, false };	// 在该边有一个以上法向时，是否处理这条边。当这条边有相邻面片才需要处理
	for (int i = 0; i < 3; ++i)
		if (adj_face_idx[i] >= 0)
		{
			adj_edge_idx[i] = adj_face_idx[i] & 0x3;
			adj_face_idx[i] = adj_face_idx[i] >> 2;
			//handle[i] = true;
		}

	int n_count[3];
	n_count[0] = triangleListD[triangleIdx].nc[0];
	n_count[1] = triangleListD[triangleIdx].nc[1];
	n_count[2] = triangleListD[triangleIdx].nc[2];
	//printf("ncount = (%d, %d, %d): triangleIdx = %d\n", n_count[0], n_count[1], n_count[2], triangleIdx);

	float *p_x  = &triangleCtrlPointD[m_ * triangleIdx];
	float *p_y  = &triangleCtrlPointD[m_ * (f + triangleIdx)];
	float *p_z  = &triangleCtrlPointD[m_ * (f * 2 + triangleIdx)];

	int edge_ctrlpoint_idx[6] = { 5, 2,			1, 3,		7, 8 };				// 依次处理的边控制顶点的序号(0,1号属于0号边， 2,3号属于1号边， 4,5号属于2号边)
	int corner_ctrlpoint_idx[6] = { 9, 0,		0, 6,		6, 9 };				// 上面每个边控制顶点对应的角控制顶点序号(0,1号属于0号边， 2,3号属于1号边， 4,5号属于2号边)
	int oppo_corner_ctrlpoint_idx[6] = { 0, 9,	6, 0,		9, 6 };				// 上面每个角控制顶点所在边的另一个角控制顶点编号(0,1号属于0号边， 2,3号属于1号边， 4,5号属于2号边)
	//int adj_corner_ctrlpoint_idx[3][2] = { { 0, 9 }, { 6, 0 }, { 9, 6 } };		// 相邻三角形0, 1, 2号边上的控制顶点编号（仅有角点，没有边点）
	//int adj_edge_ctrlpoint_idx[3][2] = { { 2, 5 }, { 3, 1 }, { 8, 7 } };		// 相邻三角形0, 1, 2号边上的控制顶点编号（仅有边点，没有角点）
	int adjust_normal_idx[6] = { 2, 0,		0, 1,		1, 2 };
	int adj_corner_ctrlpoint_idx[3][2] = { { 0, 2 }, { 1, 0 }, { 2, 1 } };		// 相邻三角形0, 1, 2号边上的控制顶点编号（仅有角点，没有边点）

	//const float ZERO = 10e-6;
	float3 delta = make_float3(0.0f, 0.0f, 0.0f), sum = make_float3(0.0f, 0.0f, 0.0f);
	// 六个边点，按0 1 2号边的顺序处理，即5, 2, 1, 3, 7, 8号控制顶点
	//printf("for开始, triangleIdx = %d\n", triangleIdx);
	for (int i = 0; i < 6; ++i)
	{
		float3 v_ctrlpoint_corner = make_float3(*(p_x + corner_ctrlpoint_idx[i]), *(p_y + corner_ctrlpoint_idx[i]), *(p_z + corner_ctrlpoint_idx[i]));
		float3 v_ctrlpoint_corner_oppo = make_float3(*(p_x + oppo_corner_ctrlpoint_idx[i]), *(p_y + oppo_corner_ctrlpoint_idx[i]), *(p_z + oppo_corner_ctrlpoint_idx[i]));
		float3 v01 = v_ctrlpoint_corner_oppo - v_ctrlpoint_corner;
		float3 v_mid = 0.5 * (v_ctrlpoint_corner + v_ctrlpoint_corner_oppo);
		//float3 n_ctrlpoint_corner = make_float3(*(pn_x + corner_ctrlpoint_idx[i]), *(pn_y + corner_ctrlpoint_idx[i]), *(pn_z + corner_ctrlpoint_idx[i]));
		float3 n_ctrlpoint_corner = triangleListD[triangleIdx].n_adj[adjust_normal_idx[i]];
		normalize(n_ctrlpoint_corner);

		// p 是要处理的边控制顶点
		float3 p = make_float3(*(p_x + edge_ctrlpoint_idx[i]), *(p_y + edge_ctrlpoint_idx[i]), *(p_z + edge_ctrlpoint_idx[i]));
		if (n_count[i / 2] < 2)		// 该条边只有一个法向
		{
			//if (adj_face_idx[i / 2] >= 0)		// 只有当这条边的另一侧有面片时才会处理
			//{
				float3 result = p - ((p - v_ctrlpoint_corner) * n_ctrlpoint_corner) * n_ctrlpoint_corner;

#ifdef RE_LENGTH
				float len0 = length(result);
				float3 result_vector = result - v_ctrlpoint_corner;
				float l_origin = length(p - v_ctrlpoint_corner);
				float l_current = length(result_vector);
				result_vector *= l_origin / l_current;
				result = v_ctrlpoint_corner + result_vector;
				float len1 = length(result);
				printf("delta_leng_1_normal = %f\n", len1 - len0);
#endif

				delta += (result - p);

				*(p_x + edge_ctrlpoint_idx[i]) = result.x;
				*(p_y + edge_ctrlpoint_idx[i]) = result.y;
				*(p_z + edge_ctrlpoint_idx[i]) = result.z;

				sum += result;
			//}
			//printf("only one : result_%d = (%f, %f, %f)\n", edge_ctrlpoint_idx[i], result.x, result.y, result.z);
		}
		//else if (handle[i / 2])		// 该条边有一个以上法向，且需要处理
		else							// 该条边有一个以上法向
		{
			float3 n1 = triangleListD[adj_face_idx[i / 2]].n_adj[adj_corner_ctrlpoint_idx[adj_edge_idx[i / 2]][i % 2]];
			//printf("else开始, triangleIdx = %d, adj_face = %d, cp = %d, n1 = (%f, %f, %f)\n", triangleIdx, adj_face_idx[i / 2], edge_ctrlpoint_idx[i], n1.x, n1.y, n1.z);
			normalize(n1);
			//if (use_pn)
			//{
				float3 n_ave = cross(n_ctrlpoint_corner, n1);
				normalize(n_ave);
				//printf("t = %d, n_cross = %f, %f, %f\n", triangleIdx, n_ave.x, n_ave.y, n_ave.z);
				//float3 result = v_ctrlpoint_corner + v01 * n_ave * 0.333333 * n_ave;				// 原始的pn尖锐边算法，将1/3点往法向上投影，效果不佳
				float3 result = v_ctrlpoint_corner + ((p - v_ctrlpoint_corner) * n_ave) * n_ave;	// 由我的算法改良而来，将差的控制顶点往法向上投影，效果很好
#ifdef RE_LENGTH
				float len0 = length(result);
				float3 result_vector = result - v_ctrlpoint_corner;
				float l_origin = length(p - v_ctrlpoint_corner);
				float l_current = length(result_vector);
				result_vector *= l_origin / l_current;
				result = v_ctrlpoint_corner + result_vector;
				float len1 = length(result);
				printf("delta_leng_pn = %f\n", len1 - len0);
#endif
				delta += (result - p);
				*(p_x + edge_ctrlpoint_idx[i]) = result.x;
				*(p_y + edge_ctrlpoint_idx[i]) = result.y;
				*(p_z + edge_ctrlpoint_idx[i]) = result.z;
				sum += result;
				//printf("2 : result_%d = (%f, %f, %f)\n", edge_ctrlpoint_idx[i], result.x, result.y, result.z);
			//}
			//else
			//{
				//float t0 = 1.2345f, t1 = 2.3456f;
				//float3 center0, center1;
				//bool t0_exist = false, t1_exist = false;
				//if (fabs(n_ctrlpoint_corner * v01) > ZERO)
				//{
					//t0 = (v_mid - v_ctrlpoint_corner) * v01 / (n_ctrlpoint_corner * v01);
					//center0 = v_ctrlpoint_corner + t0 * n_ctrlpoint_corner;
					//t0_exist = true;
					////if (triangleIdx == 10 && i == 5)
					////{
						////printf("n0 = (%f, %f, %f), t0 = %f, center0 = (%f, %f, %f)\n",
								////n_ctrlpoint_corner.x, n_ctrlpoint_corner.y, n_ctrlpoint_corner.z, t0, center0.x, center0.y, center0.z);
					////}
				//}
				//if (fabs(n1 * v01) > ZERO)
				//{
					//t1 = (v_mid - v_ctrlpoint_corner) * v01 / (n1 * v01);
					//center1 = v_ctrlpoint_corner + t1 * n1;
					//t1_exist = true;
					////if (triangleIdx == 10 && i == 5)
					////{
						////printf("n1 = (%f, %f, %f), t1 = %f, center1 = (%f, %f, %f)\n",
								////n1.x, n1.y, n1.z, t1, center1.x, center1.y, center1.z);
					////}
				//}

				////printf("t0 = %f, t1 = %f, triangleIdx = %d, cp = %d\n", t0, t1, triangleIdx, edge_ctrlpoint_idx[i]);
				//float3 center_mid;
				//if (t0_exist && t1_exist)	// 当前三角形和相邻三角形都不精确
				//{
					//float3 center_delta = center0 - center1;
					//float t = (v_ctrlpoint_corner - center0) * center_delta / (center_delta * center_delta);
					//center_mid = center0 + t * center_delta;


					//float3 rad0 = v_ctrlpoint_corner - center0;
					//float r0 = sqrt(rad0.x * rad0.x + rad0.y * rad0.y + rad0.z * rad0.z);
					//float3 rad1 = v_ctrlpoint_corner - center1;
					//float r1 = sqrt(rad1.x * rad1.x + rad1.y * rad1.y + rad1.z * rad1.z);
					////printf("都不精确, 三角形=%d, cp=%d, t = %f, r0 = %f, r1 = %f\n", triangleIdx, edge_ctrlpoint_idx[i], t, r0, r1);
				//}
				//else if (t0_exist)			// 当前三角形不精确，相邻三角形精确
				//{
					//float t = (v_ctrlpoint_corner - center0) * n1 / (n1 * n1);
					//center_mid = center0 + t * n1;
					////printf("当前三角形不精确，相邻三角形精确, 三角形=%d, cp=%d, n1 = (%f, %f, %f), t = %f\n", triangleIdx, edge_ctrlpoint_idx[i], n1.x, n1.y, n1.z, t);
				//}
				//else if (t1_exist)			// 当前三角形精确，相邻三角形不精确
				//{
					//float t = (v_ctrlpoint_corner - center1) * n_ctrlpoint_corner / (n_ctrlpoint_corner * n_ctrlpoint_corner);
					//center_mid = center1 + t * n_ctrlpoint_corner;
					////printf("当前三角形精确，相邻三角形不精确, 三角形=%d, cp=%d, t = %f\n", triangleIdx, edge_ctrlpoint_idx[i], t);
				//}
				//else						// 当前三角形和相邻三角形都精确
				//{
					////printf("两个都精确, 三角形=%d, cp=%d\n", triangleIdx, edge_ctrlpoint_idx[i]);
					//continue;
				//}
				//float3 n_ave = v_ctrlpoint_corner - center_mid;
				//normalize(n_ave);
				//float3 result = p - ((p - v_ctrlpoint_corner) * n_ave) * n_ave;
				////printf("t = %d, n_ave = %f, %f, %f\tp = %f, %f, %f\t, result=%f, %f, %f\n", triangleIdx, n_ave.x, n_ave.y, n_ave.z, p.x, p.y, p.z, result.x, result.y, result.z);
//#ifdef RE_LENGTH
				//float len0 = length(result);
				//float3 result_vector = result - v_ctrlpoint_corner;
				//float l_origin = length(p - v_ctrlpoint_corner);
				//float l_current = length(result_vector);
				//result_vector *= l_origin / l_current;
				//result = v_ctrlpoint_corner + result_vector;
				//float len1 = length(result);
				//printf("delta_leng_my = %f\n", len1 - len0);
//#endif
				//delta += (result - p);
				//*(p_x + edge_ctrlpoint_idx[i]) = result.x;
				//*(p_y + edge_ctrlpoint_idx[i]) = result.y;
				//*(p_z + edge_ctrlpoint_idx[i]) = result.z;
				//sum += result;


				//float3 n_pn = cross(n_ctrlpoint_corner, n1);
				//normalize(n_pn);
				//float3 result_pn = v_ctrlpoint_corner + ((p - v_ctrlpoint_corner) * n_pn) * n_pn;

				//float3 del = result_pn - result;
				//float dot = n_ave * n_pn;
				////printf("del = %f, %f, %f\t\tdot = %f\n", del.x, del.y, del.z, dot);



				////printf("2 : result_%d = (%f, %f, %f)\n", edge_ctrlpoint_idx[i], result.x, result.y, result.z);
			//}
		}
	}

	// 中间控制顶点，即4号控制顶点
#ifdef LESS_THAN_2
	if (n_count[0] < 2 && n_count[1] < 2 && n_count[2] < 2)
#endif
	{
		float3 p = make_float3(*(p_x + 4), *(p_y + 4), *(p_z + 4));

		/******** 平均顶点位置，PN-Triangle方法 *********/
		//sum *= 1.0 / 6;
		//float3 result = sum + (sum - p) * 0.5;

		/******** 平均delta *********/
		delta *= center_factor / 6;
		//delta *= 1.5 / 6;
		float3 result = p + delta;

		/******** 写结果 *********/
		*(p_x + 4) = result.x;
		*(p_y + 4) = result.y;
		*(p_z + 4) = result.z;
		//printf("result_4 = (%f, %f, %f)\n", result.x, result.y, result.z);
	}
}
#else
__global__ void	move(TriangleD *triangleListD, float *triangleCtrlPointD, float *triangleNormalCtrlPointD, int m_, int f)
{
	int triangleIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if (triangleIdx >= f)
		return;

	float *p_x = &triangleCtrlPointD[m_ * triangleIdx];
	float *p_y = &triangleCtrlPointD[m_ * (f + triangleIdx)];
	float *p_z = &triangleCtrlPointD[m_ * (f * 2 + triangleIdx)];
	float *pn_x = &triangleNormalCtrlPointD[m_ * triangleIdx];
	float *pn_y = &triangleNormalCtrlPointD[m_ * (f + triangleIdx)];
	float *pn_z = &triangleNormalCtrlPointD[m_ * (f * 2 + triangleIdx)];

	/******* 点1 *******/
/*#define MOVE1*/ 	// MOVE1被定义表示永远使用原始三角形的信息进行调整，理论上是错误的，只有初始情况下正确，仅供调试时用
#ifdef MOVE1
	float3 v = triangleListD[triangleIdx].v[0];
	float3 n = triangleListD[triangleIdx].n[0];
#else
	float3 v = make_float3(*p_x, *p_y, *p_z);
	float3 n = make_float3(*pn_x, *pn_y, *pn_z);
	float length = sqrt(n.x * n.x + n.y * n.y + n.z * n.z);
	normalize(n);
#endif
	float3 p = make_float3(*(p_x + 1), *(p_y + 1), *(p_z + 1));
	float3 result = p - ((p - v) * n) * n;
	float3 delta = result - p;
	/*if (threadIdx.x == 0)*/
	/*{*/
		/*printf("triangleIdx = %d\n", triangleIdx);*/
		/*printf("待投点 = (%f, %f, %f), 法向 = (%f, %f, %f), 角点 = (%f, %f, %f),\n结果 = (%f, %f, %f), 差值 = (%f, %f, %f)\n",*/
				/*p.x, p.y, p.z,			n.x, n.y, n.z,		v.x, v.y, v.z,		result.x, result.y, result.z,		delta.x, delta.y, delta.z);*/
	/*}*/
	*(p_x + 1) = result.x;
	*(p_y + 1) = result.y;
	*(p_z + 1) = result.z;
	float3 sum = result;

	// 点2
	p = make_float3(*(p_x + 2), *(p_y + 2), *(p_z + 2));
	result = p - ((p - v) * n) * n;
	*(p_x + 2) = result.x;
	*(p_y + 2) = result.y;
	*(p_z + 2) = result.z;
	sum += result;

	/******* 点3 *******/
#ifdef MOVE1
	v = triangleListD[triangleIdx].v[1];
	n = triangleListD[triangleIdx].n[1];
#else
	v = make_float3(*(p_x + 6), *(p_y + 6), *(p_z + 6));
	n = make_float3(*(pn_x + 6), *(pn_y + 6), *(pn_z + 6));
	normalize(n);
#endif
	p = make_float3(*(p_x + 3), *(p_y + 3), *(p_z + 3));
	result = p - ((p - v) * n) * n;
	*(p_x + 3) = result.x;
	*(p_y + 3) = result.y;
	*(p_z + 3) = result.z;
	sum += result;

	// 点7
	p = make_float3(*(p_x + 7), *(p_y + 7), *(p_z + 7));
	result = p - ((p - v) * n) * n;
	*(p_x + 7) = result.x;
	*(p_y + 7) = result.y;
	*(p_z + 7) = result.z;
	sum += result;

	/******* 点8 *******/
#ifdef MOVE1
	v = triangleListD[triangleIdx].v[2];
	n = triangleListD[triangleIdx].n[2];
#else
	v = make_float3(*(p_x + 9), *(p_y + 9), *(p_z + 9));
	n = make_float3(*(pn_x + 9), *(pn_y + 9), *(pn_z + 9));
	normalize(n);
#endif
	p = make_float3(*(p_x + 8), *(p_y + 8), *(p_z + 8));
	result = p - ((p - v) * n) * n;
	*(p_x + 8) = result.x;
	*(p_y + 8) = result.y;
	*(p_z + 8) = result.z;
	sum += result;

	// 点5
	p = make_float3(*(p_x + 5), *(p_y + 5), *(p_z + 5));
	result = p - ((p - v) * n) * n;
	*(p_x + 5) = result.x;
	*(p_y + 5) = result.y;
	*(p_z + 5) = result.z;
	sum += result;

	/******* 点4 *******/
	p = make_float3(*(p_x + 4), *(p_y + 4), *(p_z + 4));
	sum *= 1.0 / 6;
	result = sum + (sum - p) * 0.5;
	*(p_x + 4) = result.x;
	*(p_y + 4) = result.y;
	*(p_z + 4) = result.z;
}
#endif

float center_factor = 1.5f;

void calcTriangleCtrlPoint(bool adjust_silhouette, bool use_pn, AlgorithmType algo_type)
{
	if (algo_type == CYM)
	{
		float alpha = 1.0f, beta = 0.0f;
		/* 计算面片和法向的控制顶点*/
		hipblasStatus_t stat = hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
										  triangleCtrlPointNum_lower, triangleNum * 6, triangleCtrlPointNum + constrait_point_num,
										  &alpha,
										  matrixFittingD + matrixStartIdxFitting, triangleCtrlPointNum_lower,
										  sampleValueD, triangleCtrlPointNum + constrait_point_num,
										  &beta,
										  triangleCtrlPointD, triangleCtrlPointNum_lower);
		if (stat != HIPBLAS_STATUS_SUCCESS)
		{
			cout << "triangleCtrlPointD fail!!!!!!!!!!!!!\tstat = " << stat << endl;
			printCudaError(__FILE__, __FUNCTION__, __LINE__);
			return;
		}
	}

	// 计算每个三角片的三个用于调整控制顶点的法向
	calcAdjustNormal<<<blockNumAdjNormal, blockSizeAdjNormal, sizeof(float) * blockSizeAdjNormal * 8>>>
										(triangleListD, triangleNum,
										 order[U], order[V], order[W],
										 ctrlPointNum[U], ctrlPointNum[V], ctrlPointNum[W]);

	if (algo_type == CYM)
	{
		// 调整侧影轮廓线
		if (adjust_silhouette)
		{
			const int move_block_size = 256;
			int move_block_num = ceil(static_cast<double>(triangleNum) / move_block_size);
			//cout << "move 开始" << endl;
#ifdef NEW_MOVE
			move<<<move_block_num, move_block_size>>>(triangleListD, triangleCtrlPointD, triangle_adjacent_tableD,
					triangleCtrlPointNum_lower, triangleNum, center_factor, use_pn);
#else
			move<<<move_block_num, move_block_size>>>(triangleListD, triangleCtrlPointD, triangleCtrlPointD + 3 * triangleNum * triangleCtrlPointNum_lower, triangleCtrlPointNum_lower, triangleNum);
#endif

//hipDeviceSynchronize();

			//move<<<move_block_num, move_block_size>>>(triangleListD, triangleCtrlPointD, triangleCtrlPointD + 3 * triangleNum * triangleCtrlPointNum_lower, triangle_adjacent_tableD,
					//triangleCtrlPointNum_lower, triangleNum, center_factor);
#ifndef MORPH
			cout << "center_factor = " << center_factor << endl;
#endif
			printCudaError(__FILE__, __FUNCTION__, __LINE__);
		}
#ifdef DRAW_TRIANGULAR_CTRL_POINTS
		// 将计算好的控制顶点传回内存，仅用于调试，在最终结果上显示控制顶点，测效率时需删除
		hipMemcpy(triangular_ctrl_points, triangleCtrlPointD, sizeof(float) * 3 * triangleNum * triangleCtrlPointNum_lower, hipMemcpyDeviceToHost);
#endif
	}
	else
	{
		int blockNum = ceil(static_cast<double>(triangleNum) / 128);
		calcCtrlPoint_PN<<<blockNum, 128>>>(triangleListD, triangle_adjacent_tableD, sampleValueD_PN, triangleCtrlPointD_PN, triangleNormalCtrlPointD_PN, triangleNum, triangleCtrlPointNum_lower);
#ifdef DRAW_TRIANGULAR_CTRL_POINTS
		// 将计算好的控制顶点传回内存，仅用于调试，在最终结果上显示控制顶点，测效率时需删除
		hipMemcpy(triangular_ctrl_points, triangleCtrlPointD_PN, sizeof(float) * 3 * triangleNum * triangleCtrlPointNum_lower, hipMemcpyDeviceToHost);
#endif
	}


#ifndef MORPH
	cout << "triangleNum = " << triangleNum << endl;
#endif
	//float *test = new float[6 * triangleNum * 3];
	//hipMemcpy(test, triangleNormalCtrlPointD_PN, sizeof(float) * 6 * triangleNum * 3, hipMemcpyDeviceToHost);
	//for (int i = 0; i < triangleNum; ++i)
	//{
		//for (int j = 0; j < 6; ++j)
		//{
			////cout << i * 10 + j << ", " << (i + triangleNum) * 10 + j << ", " << (i + triangleNum * 2) * 10 + j << endl;
			//cout
				//<< test[i * 6 + j] << ", "
				//<< test[(i + triangleNum) * 6 + j] << ", "
				//<< test[(i + triangleNum * 2) * 6 + j] << endl;
		//}
		//cout << "================" << endl;
	//}

	//float *test = new float[triangleCtrlPointNum_lower * triangleNum * 6];
	//hipMemcpy(test, triangleCtrlPointD, sizeof(float) * triangleCtrlPointNum_lower * triangleNum * 6, hipMemcpyDeviceToHost);
	//float *v = test, *n = test + triangleCtrlPointNum_lower * triangleNum * 3;
	//for (int i = 0; i < triangleNum; ++i)
	////for (int i = 24; i < 25; ++i)
	//{
		//cout << "i = " << i << endl;
		//for (int j = 0; j < triangleCtrlPointNum_lower; ++j)
		//{
			////if (j != 0 && j != 6) continue;
			////cout << i * 10 + j << ", " << (i + triangleNum) * 10 + j << ", " << (i + triangleNum * 2) * 10 + j << endl;
			//cout << "\t" << j << ": " << v[i * triangleCtrlPointNum_lower + j] << ", "
				 //<< v[(i + triangleNum) * triangleCtrlPointNum_lower + j] << ", "
				 //<< v[(i + triangleNum * 2) * triangleCtrlPointNum_lower + j] << "\t";
			//double x = n[i * triangleCtrlPointNum_lower + j];
			//double y = n[(i + triangleNum) * triangleCtrlPointNum_lower + j];
			//double z = n[(i + triangleNum * 2) * triangleCtrlPointNum_lower + j];
			//double length = sqrt(x * x + y * y + z * z);
			//cout << "\t" << x / length << ", " << y / length << ", " << z / length << endl;
		//}
		//cout << "================" << endl;
	//}
	//delete []test;
}

#ifdef TRUTH
void matrixMul1_truth()
{
	float alpha = 1.0f, beta = 0.0f;
	hipblasStatus_t stat = hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
									  samplePointPerTriangle * 3, triangleCtrlPointNum, triangleCtrlPointNum, 
									  &alpha,
									  BqD_truth, samplePointPerTriangle * 3,
									  B_1D_truth, triangleCtrlPointNum,
									  &beta,
									  BBD_truth, samplePointPerTriangle * 3);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CtrlPoint_truth fail!!!!!!!!!!!!!\tstat = " << stat << endl;
		printCudaError(__FILE__, __FUNCTION__, __LINE__);
		return;
	}
}
#endif

/************************************************************************************************************/

__global__ void copy(float *RD, int u_seg, int v_seg,
					 int activeThreadNumCopy, bool firstLoad, float maxX, float maxY, float maxZ,
					 TriangleD *triangleListD, int segmentPerEdge, int f, int q,
					 float *normalPtrVBO, float *texCoordPtrVBO, float *texCoord3DPtrVBO, float *vertexPtrVBO)
{
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (globalIdx >= activeThreadNumCopy)
		return;

	int triangleIdx = globalIdx / q;
	int localIdx = globalIdx % q;

	vertexPtrVBO[globalIdx * 3 + 0] = RD[triangleIdx * q + localIdx];
	vertexPtrVBO[globalIdx * 3 + 1] = RD[(triangleIdx + f) * q + localIdx];
	vertexPtrVBO[globalIdx * 3 + 2] = RD[(triangleIdx + f * 2) * q + localIdx];

	float *ND = RD + 3 * f * q;
	normalPtrVBO[globalIdx * 3 + 0] = ND[triangleIdx * + q + localIdx];
	normalPtrVBO[globalIdx * 3 + 1] = ND[(triangleIdx + f) * + q + localIdx];
	normalPtrVBO[globalIdx * 3 + 2] = ND[(triangleIdx + f * 2) * + q + localIdx];

	if (firstLoad)
	{
		// 计算纹理坐标
		float tempFloorFloat = (sqrtf((float)(localIdx) * 8 + 9) - 3) / 2;
		int floor = rintf(tempFloorFloat);
		if ((floor * 2 + 3) * (floor * 2 + 3) != localIdx * 8 + 9)
			floor = ceilf(tempFloorFloat);
		int room = localIdx - (floor + 1) * floor / 2;

		float3 barycentric_coord;
		float3 vt0 = triangleListD[triangleIdx].vt[0];
		float3 vt1 = triangleListD[triangleIdx].vt[1];
		float3 vt2 = triangleListD[triangleIdx].vt[2];
		float u, v, w;

		barycentric_coord.x = (float)(segmentPerEdge - floor) / segmentPerEdge;
		barycentric_coord.y = (float)(floor - room) / segmentPerEdge;
		barycentric_coord.z = 1.0f - barycentric_coord.x - barycentric_coord.y;
		u = vt0.x * barycentric_coord.x + vt1.x * barycentric_coord.y + vt2.x * barycentric_coord.z;
		v = vt0.y * barycentric_coord.x + vt1.y * barycentric_coord.y + vt2.y * barycentric_coord.z;
		//w = vt0.z * barycentric_coord.x + vt1.z * barycentric_coord.y + vt2.z * barycentric_coord.z;
		w = vt0.z;

		// 这是对前8个原始三角形面片的特殊处理，因为它们的第一个顶点映射到多个纹理坐标
		if ((triangleIdx < (u_seg * v_seg * 2 - v_seg) * 8) &&		// 首先保证属于前8个原始面片
			(triangleIdx / v_seg % (u_seg * 2 - 1) == 0))			// 子面片属于原始面片产生的最开始的v_seg个面片
		{
			if (floor == 0)
				u = v = 0;
			else
			{
				float v0 = vt2.y;
				float v1 = vt1.y;
				float k = (v1 - v0) / u;
				float b = v0 * (1.0 - k);
				v = k * v + b;
			}
		}

		// 存储二维纹理坐标
		texCoordPtrVBO[globalIdx * 3 + 0] = u;
		texCoordPtrVBO[globalIdx * 3 + 1] = v;
		texCoordPtrVBO[globalIdx * 3 + 2] = w;

		// 存储三维纹理坐标
		//float minMax = maxX;
		//if (minMax > maxY)
			//minMax = maxY;
		//if (minMax > maxZ)
			//minMax = maxZ;
		////texCoord3DPtrVBO[globalIdx * 3 + 0] = vertexPtrVBO[globalIdx * 3 + 0] / maxX;
		////texCoord3DPtrVBO[globalIdx * 3 + 1] = vertexPtrVBO[globalIdx * 3 + 1] / maxY;
		////texCoord3DPtrVBO[globalIdx * 3 + 2] = vertexPtrVBO[globalIdx * 3 + 2] / maxZ;
		//texCoord3DPtrVBO[globalIdx * 3 + 0] = vertexPtrVBO[globalIdx * 3 + 0] / minMax;
		//texCoord3DPtrVBO[globalIdx * 3 + 1] = vertexPtrVBO[globalIdx * 3 + 1] / minMax;
		//texCoord3DPtrVBO[globalIdx * 3 + 2] = vertexPtrVBO[globalIdx * 3 + 2] / minMax;
	}
}

#ifdef LINE
__global__ void make_bary(TriangleD *triangleListD, float *baryPtrVBO, float *oriBaryPtrVBO, int n, int q)
{
	/***************************** 生成切割后每个顶点的重心坐标 ******************************/
	int localIdx = threadIdx.x;
	float tempFloorFloat = (sqrtf((float)localIdx * 8 + 9) - 3) / 2;
	int floor = rintf(tempFloorFloat);
	if ((floor * 2 + 3) * (floor * 2 + 3) != localIdx * 8 + 9)
		floor = ceilf(tempFloorFloat);
	int room = localIdx - (floor + 1) * floor / 2;
	float3 barycentric_coord;
	barycentric_coord.x = (float)(n - floor) / n;
	barycentric_coord.y = (float)(floor - room) / n;
	barycentric_coord.z = 1.0f - barycentric_coord.x - barycentric_coord.y;

	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	baryPtrVBO[globalIdx * 3 + 0] = (float)(n - floor) / n;
	baryPtrVBO[globalIdx * 3 + 1] = barycentric_coord.y;
	baryPtrVBO[globalIdx * 3 + 2] = barycentric_coord.z;

	/***************************** 生成切割前每个顶点的重心坐标 ******************************/
	int triangleIdx = blockIdx.x;

	// 切割后三角形三个顶点在原始三角形中的重心坐标
	float3 bary_origin0 = triangleListD[triangleIdx].bary_origin[0];
	float3 bary_origin1 = triangleListD[triangleIdx].bary_origin[1];
	float3 bary_origin2 = triangleListD[triangleIdx].bary_origin[2];

	// 当前点在原始三角形上的重心坐标
	float3 bary_origin = bary_origin0 * barycentric_coord.x + bary_origin1 * barycentric_coord.y + bary_origin2 * barycentric_coord.z;

	// 存储目前处理的采样点在原始三角片上的重心坐标
	oriBaryPtrVBO[globalIdx * 3 + 0] = bary_origin.x;
	oriBaryPtrVBO[globalIdx * 3 + 1] = bary_origin.y;
	oriBaryPtrVBO[globalIdx * 3 + 2] = bary_origin.z;
}
#endif

#ifdef TRUTH
__global__ void copy_truth(float *RD_truth,
					 int activeThreadNumCopy, bool firstLoad,
					 TriangleD *triangleListD, int segmentPerEdge, int f, int q,
					 float *normalPtrVBO_truth, float *vertexPtrVBO_truth)
{
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (globalIdx >= activeThreadNumCopy)
		return;

	int triangleIdx = globalIdx / q;
	int localIdx = globalIdx % q;

	vertexPtrVBO_truth[globalIdx * 3 + 0] = RD_truth[triangleIdx * q * 3 + localIdx];
	vertexPtrVBO_truth[globalIdx * 3 + 1] = RD_truth[(triangleIdx + f) * q * 3 + localIdx];
	vertexPtrVBO_truth[globalIdx * 3 + 2] = RD_truth[(triangleIdx + f * 2) * q * 3 + localIdx];

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////

	float ux = RD_truth[triangleIdx * q * 3 + q + localIdx];
	float uy = RD_truth[(triangleIdx + f) * q * 3 + q + localIdx];
	float uz = RD_truth[(triangleIdx + f * 2) * q * 3 + q + localIdx];

	float vx = RD_truth[triangleIdx * q * 3 + q * 2 + localIdx];
	float vy = RD_truth[(triangleIdx + f) * q * 3 + q * 2 + localIdx];
	float vz = RD_truth[(triangleIdx + f * 2) * q * 3 + q * 2 + localIdx];

	float nx = uy * vz - uz * vy;
	float ny = uz * vx - ux * vz;
	float nz = ux * vy - uy * vx;
	float l = sqrtf(nx * nx + ny * ny + nz * nz);
	nx /= l;
	ny /= l;
	nz /= l;

	normalPtrVBO_truth[globalIdx * 3 + 0] = nx;
	normalPtrVBO_truth[globalIdx * 3 + 1] = ny;
	normalPtrVBO_truth[globalIdx * 3 + 2] = nz;
}
#endif

bool registered = false;
GLuint normalVBO = 0, texCoordVBO = 0, texCoord3DVBO = 0, vertexVBO = 0;
#ifdef LINE
GLuint baryVBO = 0, oriBaryVBO = 0;
#endif
float *normalPtrVBO;							// 读写缓冲区对象所用的指针
float *texCoordPtrVBO;							// 读写缓冲区对象所用的指针
float *texCoord3DPtrVBO;						// 读写缓冲区对象所用的指针
float *vertexPtrVBO;							// 读写缓冲区对象所用的指针
#ifdef LINE
float *baryPtrVBO, *oriBaryPtrVBO;							// 读写缓冲区对象所用的指针
#endif

struct hipGraphicsResource *normalVBO_CUDA, *texCoordVBO_CUDA, *texCoord3DVBO_CUDA, *vertexVBO_CUDA;
#ifdef LINE
struct hipGraphicsResource *baryVBO_CUDA, *oriBaryVBO_CUDA;
#endif


void tessellateD(bool firstLoad, float maxX, float maxY, float maxZ, AlgorithmType algo_type)
{
	float alpha = 1.0f, beta = 0.0f;
	// 计算三角化点的坐标和法向
	hipblasStatus_t stat;
	if (algo_type == CYM)
	{
		stat = hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
										  samplePointPerTriangle, triangleNum * 6, triangleCtrlPointNum_lower,
										  &alpha,
										  BqD, samplePointPerTriangle,
										  triangleCtrlPointD, triangleCtrlPointNum_lower,
										  &beta,
										  RD, samplePointPerTriangle);
	}
	//else if (algo_type == PN_CUTTING)
	else
	{
		stat = hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
										  samplePointPerTriangle, triangleNum * 3, triangleCtrlPointNum_lower,
										  &alpha,
										  BqD, samplePointPerTriangle,
										  triangleCtrlPointD_PN, triangleCtrlPointNum_lower,
										  &beta,
										  RD, samplePointPerTriangle);
		stat = hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
										  samplePointPerTriangle, triangleNum * 3, 6,
										  &alpha,
										  BqD_PN, samplePointPerTriangle,
										  triangleNormalCtrlPointD_PN, 6,
										  &beta,
										  RD + samplePointPerTriangle * triangleNum * 3, samplePointPerTriangle);
	}
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "RD fail!!!!!!!!!!!!!\tstat = " << stat << "\t\t";
		printCudaError(__FILE__, __FUNCTION__, __LINE__);
		cout << endl;
		return;
	}

	hipGraphicsMapResources(1, &normalVBO_CUDA, 0);
	hipGraphicsMapResources(1, &texCoordVBO_CUDA, 0);
	hipGraphicsMapResources(1, &texCoord3DVBO_CUDA, 0);
	hipGraphicsMapResources(1, &vertexVBO_CUDA, 0);
#ifdef LINE
	hipGraphicsMapResources(1, &baryVBO_CUDA, 0);
	hipGraphicsMapResources(1, &oriBaryVBO_CUDA, 0);
#endif
	//size_t size2 = sizeof(float) * samplePointPerTriangle * triangleNum * 2;
	size_t size3 = sizeof(float) * samplePointPerTriangle * triangleNum * 3;
	hipGraphicsResourceGetMappedPointer((void**)&normalPtrVBO, &size3, normalVBO_CUDA);
	hipGraphicsResourceGetMappedPointer((void**)&texCoordPtrVBO, &size3, texCoordVBO_CUDA);
	hipGraphicsResourceGetMappedPointer((void**)&texCoord3DPtrVBO, &size3, texCoord3DVBO_CUDA);
	hipGraphicsResourceGetMappedPointer((void**)&vertexPtrVBO, &size3, vertexVBO_CUDA);
#ifdef LINE
	hipGraphicsResourceGetMappedPointer((void**)&baryPtrVBO, &size3, baryVBO_CUDA);
	hipGraphicsResourceGetMappedPointer((void**)&oriBaryPtrVBO, &size3, oriBaryVBO_CUDA);
#endif

	copy<<<blockNumCopy, blockSizeCopy>>>(RD, u_seg, v_seg,
										  activeThreadNumCopy, firstLoad, maxX, maxY, maxZ, triangleListD, segmentPerEdge, triangleNum, samplePointPerTriangle,
										  normalPtrVBO, texCoordPtrVBO, texCoord3DPtrVBO, vertexPtrVBO);

#ifdef LINE
	make_bary<<<triangleNum, samplePointPerTriangle>>>(triangleListD, baryPtrVBO, oriBaryPtrVBO, segmentPerEdge, samplePointPerTriangle);
#endif

	hipGraphicsUnmapResources(1, &normalVBO_CUDA, 0);
	hipGraphicsUnmapResources(1, &texCoordVBO_CUDA, 0);
	hipGraphicsUnmapResources(1, &texCoord3DVBO_CUDA, 0);
	hipGraphicsUnmapResources(1, &vertexVBO_CUDA, 0);
#ifdef LINE
	hipGraphicsUnmapResources(1, &baryVBO_CUDA, 0);
	hipGraphicsUnmapResources(1, &oriBaryVBO_CUDA, 0);
#endif
}

//#ifdef TRUTH
GLuint normalVBO_truth = 0, vertexVBO_truth = 0;
float *normalPtrVBO_truth;						// 读写缓冲区对象所用的指针
float *vertexPtrVBO_truth;						// 读写缓冲区对象所用的指针

struct hipGraphicsResource* normalVBO_CUDA_truth;
struct hipGraphicsResource* vertexVBO_CUDA_truth;

//double vertex_error_ave_max = 0.0, vertex_error_max_max = 0.0;
//double normal_error_ave_max = 0.0, normal_error_max_max = 0.0;

//int triangleCoord(int floor, int room)
//{
	//return (1 + floor) * floor / 2 + room;
//}

//__global__ void my_to_truth(int f, int q, int point_per_real_face_u, int point_per_real_face_v,
		//float *myV, float *realV, float *parameterD, int *my_to_truth_tableD, int *belongs_to_originD)
//{
	//int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
	//int triangleIdx = globalIdx / q;
	//if (triangleIdx >= f)
		//return;
	//realV += belongs_to_originD[triangleIdx] * point_per_real_face_u * point_per_real_face_v * 3;

	//float u = parameterD[globalIdx * 3];
	//float v = parameterD[globalIdx * 3 + 1];
	//int u_pre = (int)(u * (point_per_real_face_u - 1));
	//int v_pre = (int)(v * (point_per_real_face_v - 1));
	//if (u_pre < 0)
		//u_pre = 0;
	//if (v_pre < 0)
		//v_pre = 0;
	//int u_next = u_pre + 1;
	//int v_next = v_pre + 1;
	//if (u_next > point_per_real_face_u - 1)
		//u_next = point_per_real_face_u - 1;
	//if (v_next > point_per_real_face_v - 1)
		//v_next = point_per_real_face_v - 1;
	////if (blockIdx.x == 0)
	////{
		////printf("u, v = %f, %f\n", u, v);
	////}

	//float dist_min = 999999, idx_min = -1;
	//// 编号u，v的点
		//float dx = myV[globalIdx * 3] - realV[(u_pre * point_per_real_face_v + v_pre) * 3];
		//float dy = myV[globalIdx * 3 + 1] - realV[(u_pre * point_per_real_face_v + v_pre) * 3 + 1];
		//float dz = myV[globalIdx * 3 + 2] - realV[(u_pre * point_per_real_face_v + v_pre) * 3 + 2];
		//float dist = sqrt(dx * dx + dy * dy + dz * dz);
		//if (dist < dist_min)
		//{
			//dist_min = dist;
			//idx_min = u_pre * point_per_real_face_v + v_pre;
		//}

	//// 编号u+1，v的点
		//dx = myV[globalIdx * 3] - realV[(u_next * point_per_real_face_v + v_pre) * 3];
		//dy = myV[globalIdx * 3 + 1] - realV[(u_next * point_per_real_face_v + v_pre) * 3 + 1];
		//dz = myV[globalIdx * 3 + 2] - realV[(u_next * point_per_real_face_v + v_pre) * 3 + 2];
		//dist = sqrt(dx * dx + dy * dy + dz * dz);
		//if (dist < dist_min)
		//{
			//dist_min = dist;
			//idx_min = u_next * point_per_real_face_v + v_pre;
		//}

	//// 编号u，v+1的点
		//dx = myV[globalIdx * 3] - realV[(u_pre * point_per_real_face_v + v_next) * 3];
		//dy = myV[globalIdx * 3 + 1] - realV[(u_pre * point_per_real_face_v + v_next) * 3 + 1];
		//dz = myV[globalIdx * 3 + 2] - realV[(u_pre * point_per_real_face_v + v_next) * 3 + 2];
		//dist = sqrt(dx * dx + dy * dy + dz * dz);
		//if (dist < dist_min)
		//{
			//dist_min = dist;
			//idx_min = u_pre * point_per_real_face_v + v_next;
		//}

	//// 编号u+1，v+1的点
		//dx = myV[globalIdx * 3] - realV[(u_next * point_per_real_face_v + v_next) * 3];
		//dy = myV[globalIdx * 3 + 1] - realV[(u_next * point_per_real_face_v + v_next) * 3 + 1];
		//dz = myV[globalIdx * 3 + 2] - realV[(u_next * point_per_real_face_v + v_next) * 3 + 2];
		//dist = sqrt(dx * dx + dy * dy + dz * dz);
		//if (dist < dist_min)
		//{
			//dist_min = dist;
			//idx_min = u_next * point_per_real_face_v + v_next;
		//}
	//my_to_truth_tableD[globalIdx] = idx_min + belongs_to_originD[triangleIdx] * point_per_real_face_u * point_per_real_face_v;
//}

__global__ void deformTeapot(float3 *vertexParamListD_teapot, float3 *normalParamListD_teapot,
		int activeThreadNum,
		float *normalPtrVBO_truth, float *vertexPtrVBO_truth,
		int orderU, int orderV, int orderW,
		int ctrlPointNumU, int ctrlPointNumV, int ctrlPointNumW)
{
	// u, v, w 表示经过重心坐标插值之后的采样点的x, y, z分量
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= activeThreadNum)
		return;
	float u = vertexParamListD_teapot[i].x;
	float v = vertexParamListD_teapot[i].y;
	float w = vertexParamListD_teapot[i].z;

	// u, v, w方向节点区间数量
	int knot_interval_count_u = orderU + ctrlPointNumU - (orderU - 1) * 2 - 1;
	int knot_interval_count_v = orderV + ctrlPointNumV - (orderV - 1) * 2 - 1;
	int knot_interval_count_w = orderW + ctrlPointNumW - (orderW - 1) * 2 - 1;

	// 预先将其值设为最大，将末端点归入最后一段 
	int left_idx_u = orderU - 1 + knot_interval_count_u - 1;
	int left_idx_v = orderV - 1 + knot_interval_count_v - 1;
	int left_idx_w = orderW - 1 + knot_interval_count_w - 1;

	// 沿 U 方向查找当前点所在的节点区间 
	for (int ii = orderU - 1; ii <= orderU - 1 + knot_interval_count_u - 1; ++ii)
	{
		if (u >= knotListD[ii] && u < knotListD[ii + 1])
		{
			left_idx_u = ii;
			break;
		}
	}
	// 沿 V 方向查找当前点所在的节点区间 
	for (int jj = orderV - 1; jj <= orderV - 1 + knot_interval_count_v - 1; ++jj)
	{
		if (v >= knotListD[20 + jj] && v < knotListD[20 + jj + 1])
		{
			left_idx_v = jj;
			break;
		}
	}
	// 沿 W 方向查找当前点所在的节点区间 
	for (int kk = orderW - 1; kk <= orderW - 1 + knot_interval_count_w - 1; ++kk)
	{
		if (w >= knotListD[40 + kk] && w < knotListD[40 + kk + 1])
		{
			left_idx_w = kk;
			break;
		}
	}

	float tmpKnot = knotListD[left_idx_u];
	float tmpKnot1 = knotListD[left_idx_u + 1];
	float x_stride = tmpKnot1 - tmpKnot;
	u = (u - tmpKnot) / x_stride;

	tmpKnot = knotListD[20 + left_idx_v];
	tmpKnot1 = knotListD[20 + left_idx_v + 1];
	float y_stride = tmpKnot1 - tmpKnot;
	v = (v - tmpKnot) / y_stride;

	tmpKnot = knotListD[40 + left_idx_w];
	tmpKnot1 = knotListD[40 + left_idx_w + 1];
	float z_stride = tmpKnot1 - tmpKnot;
	w = (w - tmpKnot) / z_stride;

	extern __shared__ float shared_array[];
	// 算出该线程负责的采样点的 B 样条体值
	// fu 表示J_bar矩阵第一列三个元素：偏F_bar_x偏u、偏F_bar_y偏u、偏F_bar_z偏u
	// fv 表示J_bar矩阵第二列三个元素：偏F_bar_x偏v、偏F_bar_y偏v、偏F_bar_z偏v
	float3 result, fu, fv;
	BSplineVolumeValueMatrixD_combine(u, v, w, shared_array,
	left_idx_u - (orderU - 1), left_idx_v - (orderV - 1), left_idx_w - (orderW - 1),
											   orderU, orderV, orderW,
											   result, fu, fv);
	__syncthreads();

	//sampleValueD[index2c(localIdx, triangleIdx		  , m + c)] = result.x;
	//sampleValueD[index2c(localIdx, triangleIdx + f	  , m + c)] = result.y;
	//sampleValueD[index2c(localIdx, triangleIdx + f * 2, m + c)] = result.z;
	vertexPtrVBO_truth[i * 3 + 0] = result.x;
	vertexPtrVBO_truth[i * 3 + 1] = result.y;
	vertexPtrVBO_truth[i * 3 + 2] = result.z;

	//vertexPtrVBO_truth[0] = -1.0;
	//vertexPtrVBO_truth[1] = 0.0;
	//vertexPtrVBO_truth[2] = 1.0;

	//vertexPtrVBO_truth[3] = 0.0;
	//vertexPtrVBO_truth[4] = -1.0;
	//vertexPtrVBO_truth[5] = 1.0;

	//printf("%f, %f, %f = %f, %f, %f\n", u, v, w, vertexPtrVBO_truth[i * 3], vertexPtrVBO_truth[i * 3 + 1], vertexPtrVBO_truth[i * 3 + 2]);

	///////////////////////////////////////////////////////////////////////////////

	// fw 表示J_bar矩阵第三列三个元素：偏F_bar_x偏w、偏F_bar_y偏w、偏F_bar_z偏w
	float3 fw = BSplineVolumeValueMatrixDw(u, v, w, shared_array,
	left_idx_u - (orderU - 1), left_idx_v - (orderV - 1), left_idx_w - (orderW - 1),
										   orderU, orderV, orderW);
	//__syncthreads();

	u = normalParamListD_teapot[i].x;
	v = normalParamListD_teapot[i].y;
	w = normalParamListD_teapot[i].z;
	//printf("%f, %f, %f\n", u, v, w);

	//float *sampleNormalD = sampleValueD + 3 * f * (m + c);
	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第一行三个元素
	float J_bar_star_T_0 = fv.y * fw.z - fw.y * fv.z;
	float J_bar_star_T_1 = fw.y * fu.z - fu.y * fw.z;
	float J_bar_star_T_2 = fu.y * fv.z - fv.y * fu.z;
	normalPtrVBO_truth[i * 3 + 0] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;
	//sampleNormalD[index2c(localIdx, triangleIdx, m + c)] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;

	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第二行三个元素
	J_bar_star_T_0 = fv.z * fw.x - fw.z * fv.x;
	J_bar_star_T_1 = fw.z * fu.x - fu.z * fw.x;
	J_bar_star_T_2 = fu.z * fv.x - fv.z * fu.x;
	normalPtrVBO_truth[i * 3 + 1] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;
	//sampleNormalD[index2c(localIdx, triangleIdx + f, m + c)] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;

	// J_bar_star_T_[012]表示J_bar的伴随矩阵的转置(即J_bar*T)的第三行三个元素
	J_bar_star_T_0 = fv.x * fw.y - fw.x * fv.y;
	J_bar_star_T_1 = fw.x * fu.y - fu.x * fw.y;
	J_bar_star_T_2 = fu.x * fv.y - fv.x * fu.y;
	normalPtrVBO_truth[i * 3 + 2] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;
	//sampleNormalD[index2c(localIdx, triangleIdx + f * 2, m + c)] = u * J_bar_star_T_0 * x_stride + v * J_bar_star_T_1 * y_stride + w * J_bar_star_T_2 * z_stride;

	//printf("%f, %f, %f = %f, %f, %f\n", u, v, w, normalPtrVBO_truth[i * 3], normalPtrVBO_truth[i * 3 + 1], normalPtrVBO_truth[i * 3 + 2]);
}





using namespace objdata;

double color_map_vertex(const VertexCoord &v0, const VertexCoord v1, double range)
{
	return (v0 - v1).norm() / range;
}

double color_map_normal(const NormalCoord &n0, const NormalCoord &n1, double range)
{
	double result = 2 * asin((n0 - n1).norm() * 0.5) / range;
	if (result < 0)
		result = 0;
	else if (result > 1)
		result = 1;
	return result;
	//return 2 * asin((n0 - n1).norm() * 0.5) / range;
}

__device__ float power(float a, int x)
{
	float result = 1;
	for (int i = 0; i < x; ++i)
		result *= a;
	return result;
}

__device__ float B(float t, int n, int i)
{
	int factorial[] = {1, 1, 2, 6};
	double factorial_1[] = {1, 1, 0.5, 0.166666666666666666};

	return factorial[n] * factorial_1[i] * factorial_1[n - i]
			* power(t, i) * power(1 - t, n - i);
}

__global__ void calc3Dparameter(float *parameterD, float3 *parameter3D, float3 *parameterND, int total)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= total)
		return;

	float u = parameterD[idx * 3 + 0];
	float v = parameterD[idx * 3 + 1];
	int surface_id = parameterD[idx * 3 + 2];

	float3 *cp = teapot_ctrl_pointD + surface_id * 16;

	float3 result = make_float3(0, 0, 0);
	for (int i = 0; i < 4; ++i)
	{
		float3 temp_ctrlpoint = make_float3(0, 0, 0);
		for (int j = 0; j < 4; ++j)
		{
			temp_ctrlpoint += B(v, 3, j) * cp[i * 4 + j];
		}
		result += B(u, 3, i) * temp_ctrlpoint;
	}
	parameter3D[idx] = result;
	//printf("3d = %f, %f, %f\n", result.x, result.y, result.z);
	//if (parameter3D[idx].x > 1.0)
		//parameter3D[idx].x = 1.0;
	//else if (parameter3D[idx].x < -1.0)
		//parameter3D[idx].x = -1.0;

	//if (parameter3D[idx].y > 1.0)
		//parameter3D[idx].y = 1.0;
	//else if (parameter3D[idx].y < -1.0)
		//parameter3D[idx].y = -1.0;

	//if (parameter3D[idx].z > 1.0)
		//parameter3D[idx].z = 1.0;
	//else if (parameter3D[idx].z < -1.0)
		//parameter3D[idx].z = -1.0;
	//if (blockIdx.x == 0)
		//printf("%d : (%f, %f, %f) = (%f, %f, %f)\n", idx, u, v, surface_id, result.x, result.y, result.z);

	if (surface_id < 4 && u < 0.0001)				// 处理壶盖顶部的三角形面片
	{
		parameterND[idx] = make_float3(0, 0, 1);
		//printf("xiaoyu\n");
	}
	else if (surface_id < 8 && u < 0.0001)			// 处理壶身底部的三角形面片
	{
		parameterND[idx] = make_float3(0, 0, -1);
		//printf("xiaoyu2\n");
	}
	else
	{
		// 计算(u, v)点u方向导矢的值
		float3 result_u = make_float3(0, 0, 0);
		for (int j = 0; j < 4; ++j)
		{
			float3 temp_ctrlpoint = make_float3(0, 0, 0);
			for (int i = 0; i < 3; ++i)
			{
				temp_ctrlpoint += B(u, 2, i) * (cp[i * 4 + j] - cp[(i + 1) * 4 + j]);
			}
			result_u += B(v, 3, j) * temp_ctrlpoint;
		}

		// 计算(u, v)点v方向导矢的值
		float3 result_v = make_float3(0, 0, 0);
		for (int i = 0; i < 4; ++i)
		{
			float3 temp_ctrlpoint = make_float3(0, 0, 0);
			for (int j = 0; j < 3; ++j)
			{
				temp_ctrlpoint += B(v, 2, j) * (cp[i * 4 + j] - cp[i * 4 + j + 1]);
			}
			result_v += B(u, 3, i) * temp_ctrlpoint;
		}
		//Point3 normal = cross(result_u, result_v);
		float3 normal = cross(result_v, result_u);
		normalize(normal);
		parameterND[idx] = normal;
		//if (blockIdx.x == 0)
		//printf("(%f, %f, %f) = (%f, %f, %f)\n", u, v, surface_id, normal.x, normal.y, normal.z);

		//if (surface_id < 4 && u < 0.0001)
		//{
			////parameterND[idx] = make_float3(0, 0, 1);
			//printf("xiaoyu, ND = %f, %f, %f\n", parameterND[idx].x, parameterND[idx].y, parameterND[idx].z);
		//}
		//else if (surface_id < 8 && u < 0.0001)
		//{
			////parameterND[idx] = make_float3(0, 0, -1);
			//printf("xiaoyu2, ND = %f, %f, %f\n", parameterND[idx].x, parameterND[idx].y, parameterND[idx].z);
		//}
	}
}

float *texture_coord;

void tessellateD_truth(bool adjust_silhouette, bool firstLoad)
{
	hipError_t cymError;
	//cymError = hipMemcpy(my_to_truth_table, my_to_truth_tableD, sizeof(int) * samplePointPerTriangle * triangleNum, hipMemcpyDeviceToHost);
	//if (cymError)
		//cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;
	//size_t size2 = sizeof(float) * samplePointPerTriangle * triangleNum * 2;
	size_t size3 = sizeof(float) * samplePointPerTriangle * triangleNum * 3;
	//size_t size3_truth = sizeof(float) * samplePointPerTriangle * triangleNum * 3;
	cymError = hipGetLastError();
	if (cymError)
		cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;
	hipGraphicsMapResources(1, &normalVBO_CUDA, 0);
	hipGraphicsMapResources(1, &vertexVBO_CUDA, 0);
	hipGraphicsMapResources(1, &texCoordVBO_CUDA, 0);
	hipGraphicsMapResources(1, &texCoord3DVBO_CUDA, 0);
	hipGraphicsMapResources(1, &normalVBO_CUDA_truth, 0);
	hipGraphicsMapResources(1, &vertexVBO_CUDA_truth, 0);
	hipGraphicsResourceGetMappedPointer((void**)&normalPtrVBO, &size3, normalVBO_CUDA);
	hipGraphicsResourceGetMappedPointer((void**)&vertexPtrVBO, &size3, vertexVBO_CUDA);
	hipGraphicsResourceGetMappedPointer((void**)&texCoordPtrVBO, &size3, texCoordVBO_CUDA);
	hipGraphicsResourceGetMappedPointer((void**)&texCoord3DPtrVBO, &size3, texCoord3DVBO_CUDA);
	hipGraphicsResourceGetMappedPointer((void**)&normalPtrVBO_truth, &size3, normalVBO_CUDA_truth);
	hipGraphicsResourceGetMappedPointer((void**)&vertexPtrVBO_truth, &size3, vertexVBO_CUDA_truth);

	// 计算参数
	if (firstLoad)
	{
		int block_size = 128;
		int block_num = ceil(static_cast<double>(triangleNum * samplePointPerTriangle) / block_size);
		calc3Dparameter<<<block_num, block_size>>>(texCoordPtrVBO, parameter3D, parameterND, samplePointPerTriangle * triangleNum);
		cymError = hipGetLastError();
		if (cymError)
			cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;

		texture_coord = new float[size3 / sizeof(float)];
		//cout << "第一次，找对应" << endl;
		//cout << "block_num = " << block_num << endl;
		//int size = belongs_to_origin.size();
		//int *belongs_to_originD;
		//hipMalloc((void**)&belongs_to_originD, sizeof(int) * size);
		////cout << "belongsize = " << size << endl;
		//cymError = hipMemcpy(belongs_to_originD, &belongs_to_origin[0], sizeof(int) * size, hipMemcpyHostToDevice);
		//if (cymError)
			//cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;
		//block_size = 128;
		//block_num = ceil(static_cast<double>(triangleNum * samplePointPerTriangle) / block_size);
		//cout << "my_to_truth.blockNum = " << block_num << endl;
		//my_to_truth<<<block_num, block_size>>>(triangleNum, samplePointPerTriangle,
				//(u_seg + 1), (v_seg + 1), vertexPtrVBO, vertexPtrVBO_truth, parameterD,
				//my_to_truth_tableD, belongs_to_originD);
		//cymError = hipGetLastError();
		//if (cymError)
			//cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;

		//cymError = hipMemcpy(my_to_truth_table, my_to_truth_tableD, sizeof(int) * samplePointPerTriangle * triangleNum, hipMemcpyDeviceToHost);
		//if (cymError)
			//cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;


		//cout << "triangleNum = " << triangleNum << ", samplePointPerTriangle = "
			 //<< samplePointPerTriangle << endl;
		//for (int i = 0; i < samplePointPerTriangle * triangleNum; ++i)
		//{
			//cout << my_to_truth_table[i] << " ";
			//if (i % 20 == 19)
				//cout << endl;
		//}

		//int *tttt = new int[size];
		//hipMemcpy(tttt, belongs_to_originD, sizeof(int) * size, hipMemcpyDeviceToHost);
		//for (int i = 0; i < size; ++i)
		//{
			//cout << tttt[i] << " ";
			//if (i % 20 == 19)
				//cout << endl;
		//}
		//delete []tttt;
	}
	//cout << "计算参数完成" << endl;

	// 变形基准茶壶
	int block_size = 128;
	int block_num = ceil(static_cast<double>(triangleNum * samplePointPerTriangle) / block_size);
	deformTeapot<<<block_num, block_size, sizeof(float) * block_size * 13>>>
					(parameter3D, parameterND,
					triangleNum * samplePointPerTriangle,
					normalPtrVBO_truth, vertexPtrVBO_truth,
					order[U], order[V], order[W],
					ctrlPointNum[U], ctrlPointNum[V], ctrlPointNum[W]);
	cymError = hipGetLastError();
	if (cymError)
		cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;
	//cout << "基准茶壶变形完成" << endl;

	//hipMemcpy(vertexPtrVBO_truth, vertexPtrVBO, sizeof(float) * 3 * samplePointPerTriangle * 8, hipMemcpyDeviceToDevice);

	/*------------------------ 测量误差 ----------------------------*/
	/* 顶点误差 */
	float *result = new float[size3 / sizeof(float)];
	float *result_truth = new float[size3 / sizeof(float)];

	cymError = hipMemcpy(result, vertexPtrVBO, size3, hipMemcpyDeviceToHost);
	if (cymError)
		cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;
	cymError = hipMemcpy(result_truth, vertexPtrVBO_truth, size3, hipMemcpyDeviceToHost);
	if (cymError)
		cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;

	double vertex_error_ave_max = 0.0, vertex_error_max_max = 0.0;
	double normal_error_ave_max = 0.0, normal_error_max_max = 0.0;
	double error_ave = 0.0, error_max = 0.0;
	//cout << "准备进入for循环计算每个点的误差, size3 = " << size3 << ", size3 = " << size3 << endl;
	//double maxZ = -100000, minZ = 100000, minX, maxX, minY, maxY;
	for (int i = 0; i < samplePointPerTriangle * triangleNum; ++i)
	{
		double x0 = result[i * 3];
		double y0 = result[i * 3 + 1];
		double z0 = result[i * 3 + 2];
		//if (z0 > maxZ)
		//{
			//maxX = x0;
			//maxY = y0;
			//maxZ = z0;
		//}
		//if (z0 < minZ)
		//{
			//minX = x0;
			//minY = y0;
			//minZ = z0;
		//}
		//int real_idx = my_to_truth_table[i];
		//double x1 = result_truth[real_idx * 3];
		//double y1 = result_truth[real_idx * 3 + 1];
		//double z1 = result_truth[real_idx * 3 + 2];
		double x1 = result_truth[i * 3];
		double y1 = result_truth[i * 3 + 1];
		double z1 = result_truth[i * 3 + 2];
		//cout << "顶点 " << x0 << ", " << y0 << ", " << z0 <<  "====" << x1 << ", " << y1 << ", " << z1 << endl;
		double error = sqrt((x0 - x1) * (x0 - x1) + (y0 - y1) * (y0 - y1) + (z0 - z1) * (z0 - z1));
		//cout << "error = " << error << endl;
		error_ave += error;
		if (error_max < error)
			error_max = error;

		float vertex_diff = color_map_vertex(VertexCoord(x0, y0, z0), VertexCoord(x1, y1, z1), 0.04);
		texture_coord[i * 3] = vertex_diff;
		texture_coord[i * 3 + 1] = 0.5;
	}
	cout << "for循环完成" << endl;
	//cout << "maxZ = " << maxZ << ", minZ = " << minZ << endl;
	//cout << "min = " << minX << ", " << minY << ", " << minZ << endl;
	//cout << "max = " << maxX << ", " << maxY << ", " << maxZ << endl;
	hipMemcpy(texCoordPtrVBO, texture_coord, size3, hipMemcpyHostToDevice);
	/*cout << "eeeeee samplePonitPerTriangle = " << samplePointPerTriangle << endl;*/
	/*cout << "eeeeee triangleNum = " << triangleNum << endl;*/
	/*cout << "eeeeee samplePonitPerTriangle * triangleNum = " << samplePointPerTriangle * triangleNum << endl;*/

	/*cout << "eeeeee error = " << error_ave / (samplePointPerTriangle * triangleNum)*/
		 /*<< ", error_max = " << error_max << endl;*/

	if (error_ave > vertex_error_ave_max)
		vertex_error_ave_max = error_ave;
	if (error_max > vertex_error_max_max)
		vertex_error_max_max = error_max;
	if (adjust_silhouette)
		cout << "调整过，误差大" << endl;
	else
		cout << "未调整，误差小" << endl;
	cout << "eeeeee 平均顶点误差 = " << vertex_error_ave_max / (samplePointPerTriangle * triangleNum) << ", 最大顶点误差 = " << vertex_error_max_max << endl;
	cout << "error_ave = " << error_ave / (samplePointPerTriangle * triangleNum) << endl;

	//[> 体积误差 <]
	//double volume = 0.0;
	//for (int f = 0; f < triangleNum; ++f)
	//{
		//for (int i = 0; i < segmentPerEdge; ++i)
		//{
			//for (int j = 0; j <= i; ++j)
			//{
				//// smooth FFD算法结果
				//double v0x = result[samplePointPerTriangle * 3 * f + triangleCoord(i, j) * 3 + 0];
				//double v0y = result[samplePointPerTriangle * 3 * f + triangleCoord(i, j) * 3 + 1];
				//double v0z = result[samplePointPerTriangle * 3 * f + triangleCoord(i, j) * 3 + 2];
				//double v1x = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j) * 3 + 0];
				//double v1y = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j) * 3 + 1];
				//double v1z = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j) * 3 + 2];
				//double v2x = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j + 1) * 3 + 0];
				//double v2y = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j + 1) * 3 + 1];
				//double v2z = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j + 1) * 3 + 2];
				//volume += (v0z + v1z + v2z) * ((v1x - v0x) * (v2y - v0y) - (v2x - v0x) * (v1y - v0y));
				//if (i < segmentPerEdge - 1)
				//{
					//double v0x = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j) * 3 + 0];
					//double v0y = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j) * 3 + 1];
					//double v0z = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j) * 3 + 2];
					//double v1x = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 2, j + 1) * 3 + 0];
					//double v1y = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 2, j + 1) * 3 + 1];
					//double v1z = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 2, j + 1) * 3 + 2];
					//double v2x = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j + 1) * 3 + 0];
					//double v2y = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j + 1) * 3 + 1];
					//double v2z = result[samplePointPerTriangle * 3 * f + triangleCoord(i + 1, j + 1) * 3 + 2];
					//volume += (v0z + v1z + v2z) * ((v1x - v0x) * (v2y - v0y) - (v2x - v0x) * (v1y - v0y));
				//}
			//}
		//}
	//}
	//volume /= 6;
	//double volume_truth = 0.0;
	//for (vector<int>::size_type i = 0; i < teapotFaceList.size() / 3; ++i)
	//{
		//int id0 = teapotFaceList[i * 3];
		//int id1 = teapotFaceList[i * 3 + 1];
		//int id2 = teapotFaceList[i * 3 + 2];
		//double v0x = result_truth[id0 * 3];
		//double v0y = result_truth[id0 * 3 + 1];
		//double v0z = result_truth[id0 * 3 + 2];
		//double v1x = result_truth[id1 * 3];
		//double v1y = result_truth[id1 * 3 + 1];
		//double v1z = result_truth[id1 * 3 + 2];
		//double v2x = result_truth[id2 * 3];
		//double v2y = result_truth[id2 * 3 + 1];
		//double v2z = result_truth[id2 * 3 + 2];
		//volume_truth += (v0z + v1z + v2z) * ((v1x - v0x) * (v2y - v0y) - (v2x - v0x) * (v1y - v0y));
	//}
	//volume_truth /= 6;
	//cout << "eeeeee 近似体积 = " << volume << ", 真实体积 = " << volume_truth << endl;
	//cout << "eeeeee 体积误差 = " << volume - volume_truth << ", 误差率 = " << fabs(volume - volume_truth) / volume_truth << endl;

	/* 法向误差 */
	cymError = hipMemcpy(result, normalPtrVBO, size3, hipMemcpyDeviceToHost);
	if (cymError)
		cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;
	cymError = hipMemcpy(result_truth, normalPtrVBO_truth, size3, hipMemcpyDeviceToHost);
	if (cymError)
		cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;

	error_ave = 0.0, error_max = 0.0;
	float x0_max, y0_max, z0_max, x1_max, y1_max, z1_max;
	const float PI = 3.14159265358979;
	for (int i = 0; i < samplePointPerTriangle * triangleNum; ++i)
	{
		double x0 = result[i * 3];
		double y0 = result[i * 3 + 1];
		double z0 = result[i * 3 + 2];
		//int real_idx = my_to_truth_table[i];
		//double x1 = result_truth[real_idx * 3];
		//double y1 = result_truth[real_idx * 3 + 1];
		//double z1 = result_truth[real_idx * 3 + 2];
		double x1 = result_truth[i * 3];
		double y1 = result_truth[i * 3 + 1];
		double z1 = result_truth[i * 3 + 2];
		//printf("(%f, %f, %f) = (%f, %f, %f)\n", x0, y0, z0, x1, y1, z1);
		double length = sqrt(x0 * x0 + y0 * y0 + z0 * z0);
		if (fabs(length) < 0.00001)
			cout << "长度异常：" << length << endl;
		x0 /= length; y0 /= length; z0 /= length;
		length = sqrt(x1 * x1 + y1 * y1 + z1 * z1);
		if (fabs(length) < 0.00001)
			cout << "长度异常2：" << length << endl;
		x1 /= length; y1 /= length; z1 /= length;
		//cout << "ori = " << x0 << ", " << y0 << ", " << z0 << "\t"
			 //<< "deform = " << x1 << ", " << y1 << ", " << z1 << endl;
		double error = sqrt((x0 - x1) * (x0 - x1) + (y0 - y1) * (y0 - y1) + (z0 - z1) * (z0 - z1));
		//cout << "before error * 0.5 = " << error * 0.5 << "\t";
		error = 2 * asin(error * 0.5);
		//error = 1 * asin(error / 1);
		error_ave += error;
		//cout << "after error = " << error << endl;
		if (error_max < error)
		{
			error_max = error;
			x0_max = x0;
			y0_max = y0;
			z0_max = z0;
			x1_max = x1;
			y1_max = y1;
			z1_max = z1;
		}
		//double normal_diff = color_map_normal(VertexCoord(x0, y0, z0), VertexCoord(x1, y1, z1), PI / 3);
		double normal_diff = color_map_normal(VertexCoord(x0, y0, z0), VertexCoord(x1, y1, z1), PI / 30);
		texture_coord[i * 3] = normal_diff;
		texture_coord[i * 3 + 1] = 0.5;
	}
	cymError = hipGetLastError();
	if (cymError)
		cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;
	hipMemcpy(texCoord3DPtrVBO, texture_coord, size3, hipMemcpyHostToDevice);
	cymError = hipGetLastError();
	if (cymError)
		cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;
	if (error_ave > normal_error_ave_max)
		normal_error_ave_max = error_ave;
	if (error_max > normal_error_max_max)
		normal_error_max_max = error_max;
	cout << "max0 = " << x0_max << ", " << y0_max << ", " << z0_max;
	cout << "\tmax1 = " << x1_max << ", " << y1_max << ", " << z1_max << endl;
	cout << "总法向误差 = " << normal_error_ave_max << endl;
	cout << "eeeeee 平均法向误差（角度） = " << normal_error_ave_max / (samplePointPerTriangle * triangleNum) * 180 / PI
		 << ", 最大法向误差（角度） = " << normal_error_max_max * 180 / PI << endl << endl;
	cout << "error_ave = " << error_ave / (samplePointPerTriangle * triangleNum) * 180 / PI << endl;

	hipGraphicsUnmapResources(1, &normalVBO_CUDA, 0);
	hipGraphicsUnmapResources(1, &vertexVBO_CUDA, 0);
	hipGraphicsUnmapResources(1, &texCoordVBO_CUDA, 0);
	hipGraphicsUnmapResources(1, &texCoord3DVBO_CUDA, 0);
	hipGraphicsUnmapResources(1, &normalVBO_CUDA_truth, 0);
	hipGraphicsUnmapResources(1, &vertexVBO_CUDA_truth, 0);
	cymError = hipGetLastError();
	if (cymError)
		cout << __FILE__ << "第" << __LINE__ << "行, 错误代码" << cymError << ": " << hipGetErrorString(cymError) << endl;

	delete []result;
	delete []result_truth;
	/*---------------------- 测量误差完成 --------------------------*/
}
//#endif

/************************************************************************************************************/

void setGLDevice()
{
	cudaGLSetGLDevice(0);
}

/* 使用缓冲区对象进行 cuda 和 OpenGL 协同工作之前，需要进行一些初始化 */
void regGLBuffer()
{
	printCudaError(__FILE__, __FUNCTION__, __LINE__);
	if (registered)
	{
		hipGraphicsUnregisterResource(normalVBO_CUDA);
		hipGraphicsUnregisterResource(texCoordVBO_CUDA);
		hipGraphicsUnregisterResource(texCoord3DVBO_CUDA);
		hipGraphicsUnregisterResource(vertexVBO_CUDA);
#ifdef LINE
		hipGraphicsUnregisterResource(baryVBO_CUDA);
		hipGraphicsUnregisterResource(oriBaryVBO_CUDA);
#endif

//#ifdef TRUTH
		hipGraphicsUnregisterResource(normalVBO_CUDA_truth);
		hipGraphicsUnregisterResource(vertexVBO_CUDA_truth);
//#endif
		registered = false;
	}
	printCudaError(__FILE__, __FUNCTION__, __LINE__);
	hipGraphicsGLRegisterBuffer(&normalVBO_CUDA, normalVBO, cudaGraphicsMapFlagsWriteDiscard);
	printCudaError(__FILE__, __FUNCTION__, __LINE__);
	hipGraphicsGLRegisterBuffer(&texCoordVBO_CUDA, texCoordVBO, cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsGLRegisterBuffer(&texCoord3DVBO_CUDA, texCoord3DVBO, cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsGLRegisterBuffer(&vertexVBO_CUDA, vertexVBO, cudaGraphicsMapFlagsWriteDiscard);
#ifdef LINE
	hipGraphicsGLRegisterBuffer(&baryVBO_CUDA, baryVBO, cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsGLRegisterBuffer(&oriBaryVBO_CUDA, oriBaryVBO, cudaGraphicsMapFlagsWriteDiscard);
#endif
//#ifdef TRUTH
	hipGraphicsGLRegisterBuffer(&normalVBO_CUDA_truth, normalVBO_truth, cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsGLRegisterBuffer(&vertexVBO_CUDA_truth, vertexVBO_truth, cudaGraphicsMapFlagsWriteDiscard);
//#endif
	registered = true;
	printCudaError(__FILE__, __FUNCTION__, __LINE__);
}

/************************************************************************************************************/

void cudaFreeNonZero(void **ptr)
{
	if (*ptr)
	{
		hipFree(*ptr);
		*ptr = 0;
	}
}

void freeTessMemD()
{
	cudaFreeNonZero((void**)&BqD);
	cudaFreeNonZero((void**)&BqD_PN);
	cudaFreeNonZero((void**)&RD);
	cudaFreeNonZero((void**)&my_to_truth_tableD);
	cudaFreeNonZero((void**)&parameter3D);
	cudaFreeNonZero((void**)&parameterND);
	delete []my_to_truth_table;
#ifdef TRUTH
	cudaFreeNonZero((void**)&BqD_truth);
	cudaFreeNonZero((void**)&BBD_truth);
	cudaFreeNonZero((void**)&RD_truth);
#endif
}

void freeModelMemD()
{
	cudaFreeNonZero((void**)&vertexParamListD);
	cudaFreeNonZero((void**)&vertexCoordListD);
	//cudaFreeNonZero((void**)&vertexParamListD_teapot);
	//cudaFreeNonZero((void**)&vertexCoordListD_teapot);

	cudaFreeNonZero((void**)&triangleListD);
	cudaFreeNonZero((void**)&sampleValueD);
	cudaFreeNonZero((void**)&sampleValueD_PN);
	cudaFreeNonZero((void**)&triangleCtrlPointD);
	cudaFreeNonZero((void**)&triangleCtrlPointD_PN);
	cudaFreeNonZero((void**)&triangleNormalCtrlPointD_PN);
	cudaFreeNonZero((void**)&triangle_adjacent_tableD);
#ifdef TRUTH
	cudaFreeNonZero((void**)&sampleValueD_truth);
	cudaFreeNonZero((void**)&B_1D_truth);
#endif

	degreeMemD = 0;
	modelMemD = 0;

#ifdef DRAW_TRIANGULAR_CTRL_POINTS
	delete []triangular_ctrl_points;
#endif

	freeTessMemD();
}

void freeMemD()
{
	if (registered)
	{
		hipGraphicsUnregisterResource(normalVBO_CUDA);
		hipGraphicsUnregisterResource(texCoordVBO_CUDA);
		hipGraphicsUnregisterResource(texCoord3DVBO_CUDA);
		hipGraphicsUnregisterResource(vertexVBO_CUDA);
#ifdef LINE
		hipGraphicsUnregisterResource(baryVBO_CUDA);
		hipGraphicsUnregisterResource(oriBaryVBO_CUDA);
#endif
//#ifdef TRUTH
		hipGraphicsUnregisterResource(normalVBO_CUDA_truth);
		hipGraphicsUnregisterResource(vertexVBO_CUDA_truth);
//#endif
		registered = false;
	}
	if (cublas_handle)
	{
		hipblasDestroy(cublas_handle);
	}
	cudaFreeNonZero((void**)&matrixFittingIdxD);
	cudaFreeNonZero((void**)&matrixFittingD);
	permanentMemD = 0;
	freeModelMemD();
}
